#include "hip/hip_runtime.h"
/*
swsharp - CUDA parallelized Smith Waterman with applying Hirschberg's and 
Ukkonen's algorithm and dynamic cell pruning.
Copyright (C) 2013 Matija Korpar, contributor Mile Šikić

This program is free software: you can redistribute it and/or modify
it under the terms of the GNU General Public License as published by
the Free Software Foundation, either version 3 of the License, or
(at your option) any later version.

This program is distributed in the hope that it will be useful,
but WITHOUT ANY WARRANTY; without even the implied warranty of
MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
GNU General Public License for more details.

You should have received a copy of the GNU General Public License
along with this program.  If not, see <http://www.gnu.org/licenses/>.

Contact the author by mkorpar@gmail.com.
*/

#include <stdio.h>
#include <stdlib.h>
#include <string.h>

#include "chain.h"
#include "constants.h"
#include "cuda_utils.h"
#include "error.h"
#include "scorer.h"
#include "thread.h"
#include "utils.h"

#include "score_database_gpu_long.h"

#define THREADS   64
#define BLOCKS    240

#define MAX_THREADS THREADS

#define INT4_ZERO make_int4(0, 0, 0, 0)
#define INT4_SCORE_MIN make_int4(SCORE_MIN, SCORE_MIN, SCORE_MIN, SCORE_MIN)

struct LongDatabase {
    int length;
    char* codes;
    int codesLen;
    size_t codesSize;
    int* lengths;
    int* starts;
};

typedef struct LongDatabaseGpu {
    int length;
    char* codes;
    int3* data;
    int2* hBus;
    int* scores;
    size_t scoresSize;
} LongDatabaseGpu;

typedef struct Context {
    int** scores; 
    int type;
    Chain** queries;
    int queriesLen;
    LongDatabase* longDatabase;
    Scorer* scorer;
    int* indexes;
    int indexesLen;
    int* cards;
    int cardsLen;
} Context;

typedef struct KernelContext {
    int* scores; 
    int type;
    Chain** queries;
    int queriesLen;
    LongDatabase* longDatabase;
    Scorer* scorer;
    int* indexes;
    int indexesLen;
    int card;
    int queriesStart;
    int queriesStep;
} KernelContext;

typedef struct Atom {
    int mch;
    int2 up;
    int4 lScr;
    int4 lAff;
    int4 rScr;
    int4 rAff;
} Atom;

static __constant__ int gapOpen_;
static __constant__ int gapExtend_;

static __constant__ int rows_;
static __constant__ int rowsPadded_; 
static __constant__ int length_;
static __constant__ int iters_;

texture<char4, 2, hipReadModeElementType> subTexture;

//******************************************************************************
// PUBLIC

extern LongDatabase* longDatabaseCreate(Chain** database, int databaseLen);

extern void longDatabaseDelete(LongDatabase* longDatabase);

extern void scoreLongDatabaseGpu(int** scores, int type, Chain* query, 
    LongDatabase* longDatabase, Scorer* scorer, int* indexes, int indexesLen, 
    int* cards, int cardsLen, Thread* thread);

extern void scoreLongDatabasesGpu(int** scores, int type, Chain** queries, 
    int queriesLen, LongDatabase* longDatabase, Scorer* scorer, int* indexes, 
    int indexesLen, int* cards, int cardsLen, Thread* thread);

//******************************************************************************

//******************************************************************************
// PRIVATE

static void scoreDatabase(int** scores, int type, Chain** queries, 
    int queriesLen, LongDatabase* longDatabase, Scorer* scorer, int* indexes, 
    int indexesLen, int* cards, int cardsLen, Thread* thread);

static void* scoreDatabaseThread(void* param);

// cpu kernels
static void* kernel(void* param);

static void kernelSingle(int* scores, int type, Chain* query,
    LongDatabase* longDatabase, LongDatabaseGpu* longDatabaseGpu, 
    Scorer* scorer);

// gpu database preparation
static LongDatabaseGpu* longDatabaseGpuCreate(LongDatabase* longDatabase,
    int* indexes, int indexesLen);

static void longDatabaseGpuDelete(LongDatabaseGpu* longDatabaseGpu);

// gpu kernels
__global__ void hwSolve(int* scores, char* codes, int2* hBus, int3* data);

__global__ void nwSolve(int* scores, char* codes, int2* hBus, int3* data);

__global__ void swSolve(int* scores, char* codes, int2* hBus, int3* data);

__device__ static int gap(int index);

__device__ void hwSolveSingle(int* scores, char* codes, int2* hBus, int3 data);

__device__ void nwSolveSingle(int* scores, char* codes, int2* hBus, int3 data);

__device__ void swSolveSingle(int* scores, char* codes, int2* hBus, int3 data);

//******************************************************************************

//******************************************************************************
// PUBLIC

//------------------------------------------------------------------------------
// CONSTRUCTOR, DESTRUCTOR

extern LongDatabase* longDatabaseCreate(Chain** database, int databaseLen) {
    
    int* lengths = (int*) malloc(databaseLen * sizeof(int));
    int* starts = (int*) malloc(databaseLen * sizeof(int));
    
    int codesLen = 0;
    for (int i = 0; i < databaseLen; ++i) {

        int n = chainGetLength(database[i]);
        
        lengths[i] = n;
        starts[i] = codesLen;
        
        codesLen += n;        
    }
    
    size_t codesSize = codesLen * sizeof(char);
    char* codes = (char*) malloc(codesSize);
    
    for (int i = 0; i < databaseLen; ++i) {
        chainCopyCodes(database[i], codes + starts[i]);      
    }
    
    LongDatabase* longDatabase = 
        (LongDatabase*) malloc(sizeof(struct LongDatabase));
    
    longDatabase->length = databaseLen;
    longDatabase->codes = codes;
    longDatabase->codesLen = codesLen;
    longDatabase->codesSize = codesSize;
    longDatabase->lengths = lengths;
    longDatabase->starts = starts;
    
    return longDatabase;
}

extern void longDatabaseDelete(LongDatabase* longDatabase) {
    
    free(longDatabase->codes);
    free(longDatabase->starts);
    free(longDatabase->lengths);
    
    free(longDatabase);
    longDatabase = NULL;
}

//------------------------------------------------------------------------------

//------------------------------------------------------------------------------
// CPU KERNELS

extern void scoreLongDatabaseGpu(int** scores, int type, Chain* query, 
    LongDatabase* longDatabase, Scorer* scorer, int* indexes, int indexesLen, 
    int* cards, int cardsLen, Thread* thread) {
    scoreDatabase(scores, type, &query, 1, longDatabase, scorer, indexes, 
        indexesLen, cards, cardsLen, thread);
}

extern void scoreLongDatabasesGpu(int** scores, int type, Chain** queries, 
    int queriesLen, LongDatabase* longDatabase, Scorer* scorer, int* indexes, 
    int indexesLen, int* cards, int cardsLen, Thread* thread) {
    scoreDatabase(scores, type, queries, queriesLen, longDatabase, scorer,
        indexes, indexesLen, cards, cardsLen, thread);
}

//------------------------------------------------------------------------------

//******************************************************************************

//******************************************************************************
// PRIVATE

//------------------------------------------------------------------------------
// DATABASE SCORING

static void scoreDatabase(int** scores, int type, Chain** queries, 
    int queriesLen, LongDatabase* longDatabase, Scorer* scorer, int* indexes, 
    int indexesLen, int* cards, int cardsLen, Thread* thread) {
    
    ASSERT(cardsLen > 0, "no GPUs available");
    
    Context* param = (Context*) malloc(sizeof(Context));
    
    param->scores = scores;
    param->type = type;
    param->queries = queries;
    param->queriesLen = queriesLen;
    param->longDatabase = longDatabase;
    param->scorer = scorer;
    param->indexes = indexes;
    param->indexesLen = indexesLen;
    param->cards = cards;
    param->cardsLen = cardsLen;
    
    if (thread == NULL) {
        scoreDatabaseThread(param);
    } else {
        threadCreate(thread, scoreDatabaseThread, (void*) param);
    }
}

static void* scoreDatabaseThread(void* param) {

    Context* context = (Context*) param;
    
    int** scores = context->scores;
    int type = context->type;
    Chain** queries = context->queries;
    int queriesLen = context->queriesLen;
    LongDatabase* longDatabase = context->longDatabase;
    Scorer* scorer = context->scorer;
    int* indexes = context->indexes;
    int indexesLen = context->indexesLen;
    int* cards = context->cards;
    int cardsLen = context->cardsLen;
    
    int databaseLen = longDatabase->length;
    
    //**************************************************************************
    // FIX INDEXES
    
    int dummyIndexes = 0;
    
    if (indexes == NULL) {
        
        indexesLen = databaseLen;
        indexes = (int*) malloc(indexesLen * sizeof(int));
        
        for (int i = 0; i < indexesLen; ++i) {
            indexes[i] = i;
        }
        
        dummyIndexes = 1;
    }
    
    //**************************************************************************
    
    //**************************************************************************
    // SOLVE MULTICARDED
    
    *scores = (int*) malloc(queriesLen * databaseLen * sizeof(int));
    
    int threadNmr = cardsLen;
    int indexesStep = indexesLen / threadNmr;
    
    Thread* threads = (Thread*) malloc((threadNmr - 1) * sizeof(Thread));
    
    KernelContext* contexts = 
        (KernelContext*) malloc(threadNmr * sizeof(KernelContext));
    
    for (int i = 0; i < threadNmr; ++i) {
    
        contexts[i].scores = *scores;
        contexts[i].type = type;
        contexts[i].queries = queries;
        contexts[i].queriesLen = queriesLen;
        contexts[i].longDatabase = longDatabase;
        contexts[i].scorer = scorer;
        contexts[i].card = cards[i];
        contexts[i].indexes = indexes;
        contexts[i].indexesLen = indexesLen;
        
        if (threadNmr < queriesLen) {
            // one query, single card
            contexts[i].queriesStart = i;
            contexts[i].queriesStep = cardsLen;
            contexts[i].indexes = indexes;
            contexts[i].indexesLen = indexesLen;
        } else {
            // one query, multiple cards
            contexts[i].queriesStart = 0;
            contexts[i].queriesStep = 1;
            
            int offset = i * indexesStep;
            contexts[i].indexes = indexes + offset;
            contexts[i].indexesLen = min(indexesStep, indexesLen - offset);
        }
    }
    
    for (int i = 0; i < threadNmr - 1; ++i) {    
        threadCreate(&threads[i], kernel, &contexts[i]);
    }
    
    kernel(&contexts[threadNmr - 1]);
    
    for (int i = 0; i < threadNmr - 1; ++i) {
        threadJoin(threads[i]);
    }
    
    //**************************************************************************

    //**************************************************************************
    // CLEAN MEMORY

    if (dummyIndexes) {
        free(indexes);    
    }

    free(threads);
    free(contexts);
    
    free(param);
    
    //**************************************************************************
    
    return NULL;
}

//------------------------------------------------------------------------------

//------------------------------------------------------------------------------
// CPU KERNELS

static void* kernel(void* param) {

    KernelContext* context = (KernelContext*) param;
    
    int* scores = context->scores;
    int type = context->type;
    Chain** queries = context->queries;
    int queriesLen = context->queriesLen;
    LongDatabase* longDatabase = context->longDatabase;
    Scorer* scorer = context->scorer;
    int* indexes = context->indexes;
    int indexesLen = context->indexesLen;
    int card = context->card;
    int queriesStart = context->queriesStart;
    int queriesStep = context->queriesStep;
    
    // set card
    int currentCard;
    CUDA_SAFE_CALL(hipGetDevice(&currentCard));
    if (currentCard != card) {
        CUDA_SAFE_CALL(hipDeviceReset());
        CUDA_SAFE_CALL(hipSetDevice(card));
    }

    // prepare gpu db
    LongDatabaseGpu* longDatabaseGpu = longDatabaseGpuCreate(longDatabase, 
        indexes, indexesLen);

    // solve
    for (int i = queriesStart; i < queriesLen; i += queriesStep) {
    
        Chain* query = queries[i];
        int offset = i * longDatabase->length;
        
        kernelSingle(scores + offset, type, query, longDatabase, 
            longDatabaseGpu, scorer);
    }
    
    longDatabaseGpuDelete(longDatabaseGpu);
    
    return NULL;
}

static void kernelSingle(int* scores, int type, Chain* query,
    LongDatabase* longDatabase, LongDatabaseGpu* longDatabaseGpu, 
    Scorer* scorer) {

    int gapOpen = scorerGetGapOpen(scorer);
    int gapExtend = scorerGetGapExtend(scorer);
    
    int length = longDatabaseGpu->length;

    //**************************************************************************
    // CREATE QUERY PROFILE
    
    int rows = chainGetLength(query);
    int rowsGpu = rows + (4 - rows % 4) % 4;
    
    size_t rowSize = rows * sizeof(char);
    char* row = (char*) malloc(rowSize);
    chainCopyCodes(query, row);

    int subLen = SCORER_MAX_CODE + 1;
    size_t subSize = rowsGpu * subLen * sizeof(char);
    char4* subCpu = (char4*) malloc(subSize);
    memset(subCpu, 0, subSize);
    for (int i = 0; i < rowsGpu / 4; ++i) {
        for (int j = 0; j < SCORER_MAX_CODE; ++j) {
            char4 scr;
            scr.x = i * 4 + 0 >= rows ? 0 : scorerScore(scorer, row[i * 4 + 0], j);
            scr.y = i * 4 + 1 >= rows ? 0 : scorerScore(scorer, row[i * 4 + 1], j);
            scr.z = i * 4 + 2 >= rows ? 0 : scorerScore(scorer, row[i * 4 + 2], j);
            scr.w = i * 4 + 3 >= rows ? 0 : scorerScore(scorer, row[i * 4 + 3], j);
            subCpu[i * subLen + j] = scr;
        }
    }
    
    hipArray* subArray; 
    CUDA_SAFE_CALL(hipMallocArray(&subArray, &subTexture.channelDesc, subLen, rowsGpu)); 
    CUDA_SAFE_CALL(hipMemcpyToArray (subArray, 0, 0, subCpu, subSize, TO_GPU));
    CUDA_SAFE_CALL(hipBindTextureToArray(subTexture, subArray));
    subTexture.addressMode[0] = hipAddressModeClamp;
    subTexture.addressMode[1] = hipAddressModeClamp;
    subTexture.filterMode = hipFilterModePoint;
    subTexture.normalized = false;

    //**************************************************************************
    
    //**************************************************************************
    // INIT GPU
    
    int iters = rowsGpu / (THREADS * 4) + (rowsGpu % (THREADS * 4) != 0);
    
    CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(rows_), &rows, sizeof(int)));
    CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(rowsPadded_), &rowsGpu, sizeof(int)));
    CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(gapOpen_), &gapOpen, sizeof(int)));
    CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(gapExtend_), &gapExtend, sizeof(int)));
    CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(length_), &length, sizeof(int)));
    CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(iters_), &iters, sizeof(int)));
    
    //**************************************************************************

    //**************************************************************************
    // SOVLE
    
    char* codes = longDatabaseGpu->codes;
    int2* hBus = longDatabaseGpu->hBus;
    int3* data = longDatabaseGpu->data;
    int* scoresGpu = longDatabaseGpu->scores;

    void (*function)(int*, char*, int2*, int3*);
    switch (type) {
    case SW_ALIGN: 
        function = swSolve;
        break;
    case NW_ALIGN: 
        function = nwSolve;
        break;
    case HW_ALIGN:
        function = hwSolve;
        break;
    default:
        ERROR("Wrong align type");
    }
    
    function<<<BLOCKS, THREADS>>>(scoresGpu, codes, hBus, data);

    size_t scoresSize = longDatabaseGpu->scoresSize;
    CUDA_SAFE_CALL(hipMemcpy(scores, scoresGpu, scoresSize, FROM_GPU));
    
    //**************************************************************************
    
    //**************************************************************************
    // CLEAN MEMORY
    
    free(subCpu);
    free(row);
    
    CUDA_SAFE_CALL(hipFreeArray(subArray));
    
    //**************************************************************************
}

//------------------------------------------------------------------------------

//------------------------------------------------------------------------------
// GPU DATABASE PREPARE

static LongDatabaseGpu* longDatabaseGpuCreate(LongDatabase* longDatabase,
    int* indexes, int indexesLen) {
    
    size_t dataSize = indexesLen * sizeof(int3);
    int3* data = (int3*) malloc(dataSize);
    
    for (int i = 0; i < indexesLen; ++i) {
        data[i].x = indexes[i];
        data[i].y = longDatabase->starts[indexes[i]];
        data[i].z = longDatabase->lengths[indexes[i]];
    }

    int3* dataGpu;
    CUDA_SAFE_CALL(hipMalloc(&dataGpu, dataSize));
    CUDA_SAFE_CALL(hipMemcpy(dataGpu, data, dataSize, TO_GPU));
    
    free(data);
    
    size_t codesSize = longDatabase->codesSize;
    char* codesGpu;
    CUDA_SAFE_CALL(hipMalloc(&codesGpu, codesSize));
    CUDA_SAFE_CALL(hipMemcpy(codesGpu, longDatabase->codes, codesSize, TO_GPU));

    size_t hBusSize = longDatabase->codesLen * sizeof(int2);
    int2* hBus;
    CUDA_SAFE_CALL(hipMalloc(&hBus, hBusSize));

    size_t scoresSize = longDatabase->length * sizeof(int);
    int* scores = (int*) malloc(scoresSize);
    int* scoresGpu;
    CUDA_SAFE_CALL(hipMalloc(&scoresGpu, scoresSize));
    
    // init scores 
    for (int i = 0; i < longDatabase->length; ++i) {
        scores[i] = NO_SCORE;
    }
    CUDA_SAFE_CALL(hipMemcpy(scoresGpu, scores, scoresSize, TO_GPU));
    
    free(scores);
    
    LongDatabaseGpu* longDatabaseGpu = 
        (LongDatabaseGpu*) malloc(sizeof(struct LongDatabaseGpu));
    
    longDatabaseGpu->length = indexesLen;
    longDatabaseGpu->codes = codesGpu;
    longDatabaseGpu->data = dataGpu;
    longDatabaseGpu->hBus = hBus;
    longDatabaseGpu->scores = scoresGpu;
    longDatabaseGpu->scoresSize = scoresSize;
    
    return longDatabaseGpu;
}

static void longDatabaseGpuDelete(LongDatabaseGpu* longDatabaseGpu) {

    CUDA_SAFE_CALL(hipFree(longDatabaseGpu->codes));
    CUDA_SAFE_CALL(hipFree(longDatabaseGpu->data));
    CUDA_SAFE_CALL(hipFree(longDatabaseGpu->hBus));
    CUDA_SAFE_CALL(hipFree(longDatabaseGpu->scores));

    free(longDatabaseGpu);
    longDatabaseGpu = NULL;
}

//------------------------------------------------------------------------------

//------------------------------------------------------------------------------
// GPU KERNELS

__global__ void hwSolve(int* scores, char* codes, int2* hBus, int3* data) {

    for (int i = blockIdx.x; i < length_; i += gridDim.x) {
        hwSolveSingle(scores, codes, hBus, data[i]);
    }
}

__global__ void nwSolve(int* scores, char* codes, int2* hBus, int3* data) {

    for (int i = blockIdx.x; i < length_; i += gridDim.x) {
        nwSolveSingle(scores, codes, hBus, data[i]);
    }
}

__global__ void swSolve(int* scores, char* codes, int2* hBus, int3* data) {

    for (int i = blockIdx.x; i < length_; i += gridDim.x) {
        swSolveSingle(scores, codes, hBus, data[i]);
    }
}

__device__ static int gap(int index) {
    return (-gapOpen_ - index * gapExtend_) * (index >= 0);
}

__device__ void hwSolveSingle(int* scores, char* codes, int2* hBus, int3 data) {

    __shared__ int scoresShr[MAX_THREADS];

    __shared__ int hBusScrShr[MAX_THREADS + 1];
    __shared__ int hBusAffShr[MAX_THREADS + 1];

    int id = data.x;
    int off = data.y;
    int cols = data.z;

    int score = SCORE_MIN;

    int width = cols * iters_ + 2 * (blockDim.x - 1);
    int col = -threadIdx.x;
    int row = threadIdx.x * 4;
    int iter = 0;
    
    Atom atom;
    atom.mch = gap(row - 1);
    atom.lScr = make_int4(gap(row), gap(row + 1), gap(row + 2), gap(row + 3));
    atom.lAff = INT4_SCORE_MIN;
    
    hBusScrShr[threadIdx.x] = 0;
    hBusAffShr[threadIdx.x] = SCORE_MIN;
    
    for (int i = 0; i < width; ++i) {
    
        int del;
        int valid = col >= 0 && row < rowsPadded_;
    
        if (valid) {
        
            if (iter != 0 && threadIdx.x == 0) {
                atom.up = hBus[off + col];
            } else {
                atom.up.x = hBusScrShr[threadIdx.x];
                atom.up.y = hBusAffShr[threadIdx.x];
            }
            
            char code = codes[off + col];
            char4 rowScores = tex2D(subTexture, code, row >> 2);
            
            del = max(atom.up.x - gapOpen_, atom.up.y - gapExtend_);
            int ins = max(atom.lScr.x - gapOpen_, atom.lAff.x - gapExtend_);
            int mch = atom.mch + rowScores.x;

            atom.rScr.x = MAX3(mch, del, ins);
            atom.rAff.x = ins;

            del = max(atom.rScr.x - gapOpen_, del - gapExtend_);
            ins = max(atom.lScr.y - gapOpen_, atom.lAff.y - gapExtend_);
            mch = atom.lScr.x + rowScores.y;

            atom.rScr.y = MAX3(mch, del, ins);
            atom.rAff.y = ins;
            
            del = max(atom.rScr.y - gapOpen_, del - gapExtend_);
            ins = max(atom.lScr.z - gapOpen_, atom.lAff.z - gapExtend_);
            mch = atom.lScr.y + rowScores.z;

            atom.rScr.z = MAX3(mch, del, ins);
            atom.rAff.z = ins;

            del = max(atom.rScr.z - gapOpen_, del - gapExtend_);
            ins = max(atom.lScr.w - gapOpen_, atom.lAff.w - gapExtend_);
            mch = atom.lScr.z + rowScores.w;

            atom.rScr.w = MAX3(mch, del, ins);
            atom.rAff.w = ins;

            if (row + 0 == rows_ - 1) score = max(score, atom.rScr.x);
            if (row + 1 == rows_ - 1) score = max(score, atom.rScr.y);
            if (row + 2 == rows_ - 1) score = max(score, atom.rScr.z);
            if (row + 3 == rows_ - 1) score = max(score, atom.rScr.w);

            atom.mch = atom.up.x;   
            VEC4_ASSIGN(atom.lScr, atom.rScr);
            VEC4_ASSIGN(atom.lAff, atom.rAff);
        }
        
        __syncthreads();

        if (valid) {
            if (iter < iters_ - 1 && threadIdx.x == blockDim.x - 1) {
                VEC2_ASSIGN(hBus[off + col], make_int2(atom.rScr.w, del));
            } else {
                hBusScrShr[threadIdx.x + 1] = atom.rScr.w;
                hBusAffShr[threadIdx.x + 1] = del;
            }
        }
        
        col++;
        
        if (col == cols) {

            col = 0;
            row += blockDim.x * 4;
            iter++;
            
            atom.mch = gap(row - 1);
            atom.lScr = make_int4(gap(row), gap(row + 1), gap(row + 2), gap(row + 3));;
            atom.lAff = INT4_SCORE_MIN;
        }
        
        __syncthreads();
    }

    // write all scores    
    scoresShr[threadIdx.x] = score;
    __syncthreads();
    
    // gather scores
    if (threadIdx.x == 0) {
    
        for (int i = 1; i < blockDim.x; ++i) {
            score = max(score, scoresShr[i]);
        }
    
        scores[id] = score;
    }
}

__device__ void nwSolveSingle(int* scores, char* codes, int2* hBus, int3 data) {

    __shared__ int scoresShr[MAX_THREADS];

    __shared__ int hBusScrShr[MAX_THREADS + 1];
    __shared__ int hBusAffShr[MAX_THREADS + 1];

    int id = data.x;
    int off = data.y;
    int cols = data.z;

    int score = SCORE_MIN;

    int width = cols * iters_ + 2 * (blockDim.x - 1);
    int col = -threadIdx.x;
    int row = threadIdx.x * 4;
    int iter = 0;
    
    Atom atom;
    atom.mch = gap(row - 1);
    atom.lScr = make_int4(gap(row), gap(row + 1), gap(row + 2), gap(row + 3));
    atom.lAff = INT4_SCORE_MIN;
    
    hBusScrShr[threadIdx.x] = gap(off);
    hBusAffShr[threadIdx.x] = SCORE_MIN;
    
    for (int i = 0; i < width; ++i) {
    
        int del;
        int valid = col >= 0 && row < rowsPadded_;
    
        if (valid) {
        
            if (iter != 0 && threadIdx.x == 0) {
                if (iter == 0) {
                   atom.up.x = gap(off);
                   atom.up.y = SCORE_MIN;
                } else {
                    atom.up = hBus[off + col];
                }
            } else {
                atom.up.x = hBusScrShr[threadIdx.x];
                atom.up.y = hBusAffShr[threadIdx.x];
            }
            
            char code = codes[off + col];
            char4 rowScores = tex2D(subTexture, code, row >> 2);
            
            del = max(atom.up.x - gapOpen_, atom.up.y - gapExtend_);
            int ins = max(atom.lScr.x - gapOpen_, atom.lAff.x - gapExtend_);
            int mch = atom.mch + rowScores.x;

            atom.rScr.x = MAX3(mch, del, ins);
            atom.rAff.x = ins;

            del = max(atom.rScr.x - gapOpen_, del - gapExtend_);
            ins = max(atom.lScr.y - gapOpen_, atom.lAff.y - gapExtend_);
            mch = atom.lScr.x + rowScores.y;

            atom.rScr.y = MAX3(mch, del, ins);
            atom.rAff.y = ins;
            
            del = max(atom.rScr.y - gapOpen_, del - gapExtend_);
            ins = max(atom.lScr.z - gapOpen_, atom.lAff.z - gapExtend_);
            mch = atom.lScr.y + rowScores.z;

            atom.rScr.z = MAX3(mch, del, ins);
            atom.rAff.z = ins;

            del = max(atom.rScr.z - gapOpen_, del - gapExtend_);
            ins = max(atom.lScr.w - gapOpen_, atom.lAff.w - gapExtend_);
            mch = atom.lScr.z + rowScores.w;

            atom.rScr.w = MAX3(mch, del, ins);
            atom.rAff.w = ins;

            atom.mch = atom.up.x;   
            VEC4_ASSIGN(atom.lScr, atom.rScr);
            VEC4_ASSIGN(atom.lAff, atom.rAff);
        }
        
        __syncthreads();

        if (valid) {
            if (iter < iters_ - 1 && threadIdx.x == blockDim.x - 1) {
                VEC2_ASSIGN(hBus[off + col], make_int2(atom.rScr.w, del));
            } else {
                hBusScrShr[threadIdx.x + 1] = atom.rScr.w;
                hBusAffShr[threadIdx.x + 1] = del;
            }
        }
        
        col++;
        
        if (col == cols) {

            if (row + 0 == rows_ - 1) score = max(score, atom.lScr.x);
            if (row + 1 == rows_ - 1) score = max(score, atom.lScr.y);
            if (row + 2 == rows_ - 1) score = max(score, atom.lScr.z);
            if (row + 3 == rows_ - 1) score = max(score, atom.lScr.w);
            
            col = 0;
            row += blockDim.x * 4;
            iter++;
            
            atom.mch = gap(row - 1);
            atom.lScr = make_int4(gap(row), gap(row + 1), gap(row + 2), gap(row + 3));;
            atom.lAff = INT4_SCORE_MIN;
        }
        
        __syncthreads();
    }

    // write all scores    
    scoresShr[threadIdx.x] = score;
    __syncthreads();
    
    // gather scores
    if (threadIdx.x == 0) {
    
        for (int i = 1; i < blockDim.x; ++i) {
            score = max(score, scoresShr[i]);
        }
    
        scores[id] = score;
    }
}

__device__ void swSolveSingle(int* scores, char* codes, int2* hBus, int3 data) {

    __shared__ int scoresShr[MAX_THREADS];

    __shared__ int hBusScrShr[MAX_THREADS + 1];
    __shared__ int hBusAffShr[MAX_THREADS + 1];

    int id = data.x;
    int off = data.y;
    int cols = data.z;

    int score = 0;
    
    int width = cols * iters_ + 2 * (blockDim.x - 1);
    int col = -threadIdx.x;
    int row = threadIdx.x * 4;
    int iter = 0;
    
    Atom atom;
    atom.mch = 0;
    atom.lScr = INT4_ZERO;
    atom.lAff = INT4_SCORE_MIN;
    
    hBusScrShr[threadIdx.x] = 0;
    hBusAffShr[threadIdx.x] = SCORE_MIN;
    
    for (int i = 0; i < width; ++i) {
    
        int del;
        int valid = col >= 0 && row < rowsPadded_;
    
        if (valid) {
        
            if (iter != 0 && threadIdx.x == 0) {
                atom.up = hBus[off + col];
            } else {
                atom.up.x = hBusScrShr[threadIdx.x];
                atom.up.y = hBusAffShr[threadIdx.x];
            }
            
            char code = codes[off + col];
            char4 rowScores = tex2D(subTexture, code, row >> 2);
            
            del = max(atom.up.x - gapOpen_, atom.up.y - gapExtend_);
            int ins = max(atom.lScr.x - gapOpen_, atom.lAff.x - gapExtend_);
            int mch = atom.mch + rowScores.x;

            atom.rScr.x = MAX4(0, mch, del, ins);
            atom.rAff.x = ins;

            del = max(atom.rScr.x - gapOpen_, del - gapExtend_);
            ins = max(atom.lScr.y - gapOpen_, atom.lAff.y - gapExtend_);
            mch = atom.lScr.x + rowScores.y;

            atom.rScr.y = MAX4(0, mch, del, ins);
            atom.rAff.y = ins;
            
            del = max(atom.rScr.y - gapOpen_, del - gapExtend_);
            ins = max(atom.lScr.z - gapOpen_, atom.lAff.z - gapExtend_);
            mch = atom.lScr.y + rowScores.z;

            atom.rScr.z = MAX4(0, mch, del, ins);
            atom.rAff.z = ins;

            del = max(atom.rScr.z - gapOpen_, del - gapExtend_);
            ins = max(atom.lScr.w - gapOpen_, atom.lAff.w - gapExtend_);
            mch = atom.lScr.z + rowScores.w;

            atom.rScr.w = MAX4(0, mch, del, ins);
            atom.rAff.w = ins;

            score = max(score, atom.rScr.x);
            score = max(score, atom.rScr.y);
            score = max(score, atom.rScr.z);
            score = max(score, atom.rScr.w);

            atom.mch = atom.up.x;   
            VEC4_ASSIGN(atom.lScr, atom.rScr);
            VEC4_ASSIGN(atom.lAff, atom.rAff);
        }
        
        __syncthreads();

        if (valid) {
            if (iter < iters_ - 1 && threadIdx.x == blockDim.x - 1) {
                VEC2_ASSIGN(hBus[off + col], make_int2(atom.rScr.w, del));
            } else {
                hBusScrShr[threadIdx.x + 1] = atom.rScr.w;
                hBusAffShr[threadIdx.x + 1] = del;
            }
        }
        
        col++;
        
        if (col == cols) {

            col = 0;
            row += blockDim.x * 4;
            iter++;
                    
            atom.mch = 0;
            atom.lScr = INT4_ZERO;
            atom.lAff = INT4_SCORE_MIN;
        }
        
        __syncthreads();
    }

    // write all scores    
    scoresShr[threadIdx.x] = score;
    __syncthreads();
    
    // gather scores
    if (threadIdx.x == 0) {
    
        for (int i = 1; i < blockDim.x; ++i) {
            score = max(score, scoresShr[i]);
        }
    
        scores[id] = score;
    }
}

//------------------------------------------------------------------------------

//******************************************************************************
