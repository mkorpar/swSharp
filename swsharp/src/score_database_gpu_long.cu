#include "hip/hip_runtime.h"
/*
swsharp - CUDA parallelized Smith Waterman with applying Hirschberg's and 
Ukkonen's algorithm and dynamic cell pruning.
Copyright (C) 2013 Matija Korpar, contributor Mile Šikić

This program is free software: you can redistribute it and/or modify
it under the terms of the GNU General Public License as published by
the Free Software Foundation, either version 3 of the License, or
(at your option) any later version.

This program is distributed in the hope that it will be useful,
but WITHOUT ANY WARRANTY; without even the implied warranty of
MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
GNU General Public License for more details.

You should have received a copy of the GNU General Public License
along with this program.  If not, see <http://www.gnu.org/licenses/>.

Contact the author by mkorpar@gmail.com.
*/

#include <stdio.h>
#include <stdlib.h>
#include <string.h>

#include "chain.h"
#include "constants.h"
#include "cuda_utils.h"
#include "error.h"
#include "scorer.h"
#include "thread.h"
#include "utils.h"

#include "score_database_gpu_long.h"

#define THREADS   128
#define BLOCKS    240

#define INT2_ZERO make_int2(0, 0)
#define SCORE4_MIN make_int4(SCORE_MIN, SCORE_MIN, SCORE_MIN, SCORE_MIN)

struct LongDatabase {
    int length;
};

typedef struct LongDatabaseGpu {

} LongDatabaseGpu;

typedef struct Context {
    int** scores; 
    int type;
    Chain** queries;
    int queriesLen;
    LongDatabase* longDatabase;
    Scorer* scorer;
    int* indexes;
    int indexesLen;
    int* cards;
    int cardsLen;
} Context;

//******************************************************************************
// PUBLIC

extern LongDatabase* longDatabaseCreate(Chain** database, int databaseLen);

extern void longDatabaseDelete(LongDatabase* longDatabase);

extern void scoreLongDatabaseGpu(int** scores, int type, Chain* query, 
    LongDatabase* longDatabase, Scorer* scorer, int* indexes, int indexesLen, 
    int* cards, int cardsLen, Thread* thread);

extern void scoreLongDatabasesGpu(int** scores, int type, Chain** queries, 
    int queriesLen, LongDatabase* longDatabase, Scorer* scorer, int* indexes, 
    int indexesLen, int* cards, int cardsLen, Thread* thread);

//******************************************************************************

//******************************************************************************
// PRIVATE

static void scoreDatabase(int** scores, int type, Chain** queries, 
    int queriesLen, LongDatabase* longDatabase, Scorer* scorer, int* indexes, 
    int indexesLen, int* cards, int cardsLen, Thread* thread);

static void* scoreDatabaseThread(void* param);

// cpu kernels
static void* kernel(void* param);

// gpu database preparation
static LongDatabaseGpu* longDatabaseGpuCreate(LongDatabase* longDatabase,
    int* indexes, int indexesLen);

static void longDatabaseGpuDelete(LongDatabaseGpu* longDatabaseGpu);

// gpu kernels



//******************************************************************************

//******************************************************************************
// PUBLIC

//------------------------------------------------------------------------------
// CONSTRUCTOR, DESTRUCTOR

extern LongDatabase* longDatabaseCreate(Chain** database, int databaseLen) {
    
    LongDatabase* longDatabase = 
        (LongDatabase*) malloc(sizeof(struct LongDatabase));
    
    longDatabase->length = databaseLen;
    
    return longDatabase;
}

extern void longDatabaseDelete(LongDatabase* longDatabase) {
    
    free(longDatabase);
    longDatabase = NULL;
}

//------------------------------------------------------------------------------

//------------------------------------------------------------------------------
// CPU KERNELS

extern void scoreLongDatabaseGpu(int** scores, int type, Chain* query, 
    LongDatabase* longDatabase, Scorer* scorer, int* indexes, int indexesLen, 
    int* cards, int cardsLen, Thread* thread) {
    scoreDatabase(scores, type, &query, 1, longDatabase, scorer, indexes, 
        indexesLen, cards, cardsLen, thread);
}

extern void scoreLongDatabasesGpu(int** scores, int type, Chain** queries, 
    int queriesLen, LongDatabase* longDatabase, Scorer* scorer, int* indexes, 
    int indexesLen, int* cards, int cardsLen, Thread* thread) {
    scoreDatabase(scores, type, queries, queriesLen, longDatabase, scorer,
        indexes, indexesLen, cards, cardsLen, thread);
}

//------------------------------------------------------------------------------

//******************************************************************************

//******************************************************************************
// PRIVATE

//------------------------------------------------------------------------------
// DATABASE SCORING

static void scoreDatabase(int** scores, int type, Chain** queries, 
    int queriesLen, LongDatabase* longDatabase, Scorer* scorer, int* indexes, 
    int indexesLen, int* cards, int cardsLen, Thread* thread) {
    
    ASSERT(cardsLen > 0, "no GPUs available");
    
    Context* param = (Context*) malloc(sizeof(Context));
    
    param->scores = scores;
    param->type = type;
    param->queries = queries;
    param->queriesLen = queriesLen;
    param->longDatabase = longDatabase;
    param->scorer = scorer;
    param->indexes = indexes;
    param->indexesLen = indexesLen;
    param->cards = cards;
    param->cardsLen = cardsLen;
    
    if (thread == NULL) {
        scoreDatabaseThread(param);
    } else {
        threadCreate(thread, scoreDatabaseThread, (void*) param);
    }
}

static void* scoreDatabaseThread(void* param) {

    Context* context = (Context*) param;
    
    int** scores = context->scores;
    int type = context->type;
    Chain** queries = context->queries;
    int queriesLen = context->queriesLen;
    LongDatabase* longDatabase = context->longDatabase;
    Scorer* scorer = context->scorer;
    int* indexes = context->indexes;
    int indexesLen = context->indexesLen;
    int* cards = context->cards;
    int cardsLen = context->cardsLen;
    
    //**************************************************************************
    // SOLVE MULTICARDED
    
    *scores = (int*) malloc(queriesLen * longDatabase->length * sizeof(int));
    
    for (int i = 0; i < longDatabase->length * queriesLen; i++) {
        (*scores)[i] = SCORE_MIN;
    }
    
    //**************************************************************************

    //**************************************************************************
    // CLEAN MEMORY

    free(param);
    
    //**************************************************************************
    
    return NULL;
}

//------------------------------------------------------------------------------

//------------------------------------------------------------------------------
// CPU KERNELS

//------------------------------------------------------------------------------

//------------------------------------------------------------------------------
// GPU DATABASE PREPARE

static LongDatabaseGpu* longDatabaseGpuCreate(LongDatabase* longDatabase,
    int* indexes, int indexesLen) {
    
    LongDatabaseGpu* longDatabaseGpu = 
        (LongDatabaseGpu*) malloc(sizeof(struct LongDatabaseGpu));
    
    return longDatabaseGpu;
}

static void longDatabaseGpuDelete(LongDatabaseGpu* longDatabaseGpu) {

    free(longDatabaseGpu);
    longDatabaseGpu = NULL;
}

//------------------------------------------------------------------------------

//------------------------------------------------------------------------------
// GPU KERNELS

//------------------------------------------------------------------------------

//******************************************************************************
