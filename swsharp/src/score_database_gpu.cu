#include "hip/hip_runtime.h"
/*
swsharp - CUDA parallelized Smith Waterman with applying Hirschberg's and 
Ukkonen's algorithm and dynamic cell pruning.
Copyright (C) 2013 Matija Korpar, contributor Mile Šikić

This program is free software: you can redistribute it and/or modify
it under the terms of the GNU General Public License as published by
the Free Software Foundation, either version 3 of the License, or
(at your option) any later version.

This program is distributed in the hope that it will be useful,
but WITHOUT ANY WARRANTY; without even the implied warranty of
MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
GNU General Public License for more details.

You should have received a copy of the GNU General Public License
along with this program.  If not, see <http://www.gnu.org/licenses/>.

Contact the author by mkorpar@gmail.com.
*/

#include <limits.h>
#include <stdio.h>
#include <stdlib.h>
#include <string.h>

#include "chain.h"
#include "constants.h"
#include "cpu_module.h"
#include "cuda_utils.h"
#include "error.h"
#include "scorer.h"
#include "score_database_gpu_long.h"
#include "score_database_gpu_short.h"
#include "thread.h"
#include "threadpool.h"
#include "utils.h"

#include "gpu_module.h"

#define MAX_SHORT_LEN       2800

#define CPU_WORKER_STEP         32
#define CPU_THREADPOOL_STEP     100

typedef struct Context {
    int** scores;
    int type;
    Chain** queries; 
    int queriesLen;
    ChainDatabaseGpu* chainDatabaseGpu;
    Scorer* scorer;
    int* indexes;
    int indexesLen;
    int* cards;
    int cardsLen;
} Context;

typedef struct ContextCpu {
    int* scores;
    int type;
    Chain** queries; 
    int queriesLen;
    Chain** database;
    int databaseLen;
    Scorer* scorer;
    int* indexes;
    int indexesLen;
    Mutex* mutex;
    int lastIndexSolved;
    int cancelled;
} ContextCpu;

typedef struct ContextWorkerCpu {
    int* scores;
    int type;
    Chain** queries; 
    int queriesLen;
    Chain** database;
    int databaseLen;
    Scorer* scorer;
    Mutex* mutex;
    int* lastQuery;
    int* lastTarget;
    int* cancelled;
} ContextWorkerCpu;

struct ChainDatabaseGpu {
    Chain** database;
    int databaseLen;
    ShortDatabase* shortDatabase;
    LongDatabase* longDatabase;
    int* longIndexes;
    int longIndexesLen;
};

//******************************************************************************
// PUBLIC

extern ChainDatabaseGpu* chainDatabaseGpuCreate(Chain** database, int databaseLen,
    int* cards, int cardsLen);

extern void chainDatabaseGpuDelete(ChainDatabaseGpu* chainDatabaseGpu);

extern void scoreDatabaseGpu(int** scores, int type, Chain* query, 
    ChainDatabaseGpu* chainDatabaseGpu, Scorer* scorer, int* indexes, 
    int indexesLen, int* cards, int cardsLen, Thread* thread);
    
extern void scoreDatabasesGpu(int** scores, int type, Chain** queries, 
    int queriesLen, ChainDatabaseGpu* chainDatabaseGpu, Scorer* scorer, 
    int* indexes, int indexesLen, int* cards, int cardsLen, Thread* thread);

//******************************************************************************

//******************************************************************************
// PRIVATE

static void scoreDatabase(int** scores, int type, Chain** queries, 
    int queriesLen, ChainDatabaseGpu* chainDatabaseGpu, Scorer* scorer, 
    int* indexes, int indexesLen, int* cards, int cardsLen, Thread* thread);
    
static void* scoreDatabaseThread(void* param);

// cpu workers
static void* scoreCpu(void* param);

static void* scoreCpuWorker(void* param);

// utils
static void filterIndexesArray(int** indexesNew, int* indexesNewLen, 
    int* indexes, int indexesLen, int minIndex, int maxIndex);

static void filterLongIndexesArray(int** longIndexesNew, int* longIndexesNewLen, 
    int* longIndexes, int longIndexesLen, int* indexes, int indexesLen,
    int maxIndex);

static int int2CmpY(const void* a_, const void* b_);

//******************************************************************************

//******************************************************************************
// PUBLIC

extern ChainDatabaseGpu* chainDatabaseGpuCreate(Chain** database, int databaseLen,
    int* cards, int cardsLen) {

    if (cardsLen == 0 || databaseLen == 0) {
        return NULL;
    }

    //**************************************************************************
    // CREATE LONG INDEXES

    int2* packed = (int2*) malloc(databaseLen * sizeof(int2));
    int packedLen = 0;

    for (int i = 0; i < databaseLen; ++i) {
    
        const int n = chainGetLength(database[i]);
        
        if (n >= MAX_SHORT_LEN) {
            packed[packedLen].x = i;
            packed[packedLen].y = n;
            packedLen++;
        }
    }

    qsort(packed, packedLen, sizeof(int2), int2CmpY);

    int longIndexesLen = packedLen;
    int* longIndexes = (int*) malloc(longIndexesLen * sizeof(int));

    for (int i = 0; i < longIndexesLen; ++i) {
        longIndexes[i] = packed[i].x;
    }

    free(packed);

    //**************************************************************************

    //**************************************************************************
    // CREATE GPU DATABASES

    ShortDatabase* shortDatabase = shortDatabaseCreate(database, databaseLen, 
        0, MAX_SHORT_LEN, cards, cardsLen);
        
    LongDatabase* longDatabase = longDatabaseCreate(database, databaseLen, 
        MAX_SHORT_LEN, INT_MAX, cards, cardsLen);
    
    //**************************************************************************

    //**************************************************************************
    // SAVE DATA

    ChainDatabaseGpu* chainDatabaseGpu = 
        (ChainDatabaseGpu*) malloc(sizeof(struct ChainDatabaseGpu));
    
    chainDatabaseGpu->database = database;
    chainDatabaseGpu->databaseLen = databaseLen;
    chainDatabaseGpu->shortDatabase = shortDatabase;
    chainDatabaseGpu->longDatabase = longDatabase;
    chainDatabaseGpu->longIndexes = longIndexes;
    chainDatabaseGpu->longIndexesLen = longIndexesLen;

    //**************************************************************************

    return chainDatabaseGpu;
}

extern void chainDatabaseGpuDelete(ChainDatabaseGpu* chainDatabaseGpu) {

    if (chainDatabaseGpu != NULL) {

        shortDatabaseDelete(chainDatabaseGpu->shortDatabase);
        longDatabaseDelete(chainDatabaseGpu->longDatabase);
        free(chainDatabaseGpu->longIndexes);

        free(chainDatabaseGpu);
    }
}

extern size_t chainDatabaseGpuMemoryConsumption(Chain** database, int databaseLen) {

    size_t mem1 = shortDatabaseGpuMemoryConsumption(database, databaseLen,
        0, MAX_SHORT_LEN);
    size_t mem2 = longDatabaseGpuMemoryConsumption(database, databaseLen,
        MAX_SHORT_LEN, INT_MAX);

    return mem1 + mem2;
}

extern void scoreDatabaseGpu(int** scores, int type, Chain* query, 
    ChainDatabaseGpu* chainDatabaseGpu, Scorer* scorer, int* indexes, 
    int indexesLen, int* cards, int cardsLen, Thread* thread) {
    scoreDatabase(scores, type, &query, 1, chainDatabaseGpu, scorer, indexes, 
        indexesLen, cards, cardsLen, thread);
}

extern void scoreDatabasesGpu(int** scores, int type, Chain** queries, 
    int queriesLen, ChainDatabaseGpu* chainDatabaseGpu, Scorer* scorer, 
    int* indexes, int indexesLen, int* cards, int cardsLen, Thread* thread) {
    scoreDatabase(scores, type, queries, queriesLen, chainDatabaseGpu, scorer,
        indexes, indexesLen, cards, cardsLen, thread);
}

//******************************************************************************
// PRIVATE

//------------------------------------------------------------------------------
// ENTRY

static void scoreDatabase(int** scores, int type, Chain** queries, 
    int queriesLen, ChainDatabaseGpu* chainDatabaseGpu, Scorer* scorer, 
    int* indexes, int indexesLen, int* cards, int cardsLen, Thread* thread) {

    ASSERT(cardsLen > 0, "no GPUs available");
    
    Context* param = (Context*) malloc(sizeof(Context));

    param->scores = scores;
    param->type = type;
    param->queries = queries;
    param->queriesLen = queriesLen;
    param->chainDatabaseGpu = chainDatabaseGpu;
    param->scorer = scorer;
    param->indexes = indexes;
    param->indexesLen = indexesLen;
    param->cards = cards;
    param->cardsLen = cardsLen;
    
    if (thread == NULL) {
        scoreDatabaseThread(param);
    } else {
        threadCreate(thread, scoreDatabaseThread, (void*) param);
    }
}
//------------------------------------------------------------------------------

//------------------------------------------------------------------------------
// SOLVE

static void* scoreDatabaseThread(void* param) {

    Context* context = (Context*) param;
    
    int** scores = context->scores;
    int type = context->type;
    Chain** queries = context->queries;
    int queriesLen = context->queriesLen;
    ChainDatabaseGpu* chainDatabaseGpu = context->chainDatabaseGpu;
    Scorer* scorer = context->scorer;
    int* indexes = context->indexes;
    int indexesLen = context->indexesLen;
    int* cards = context->cards;
    int cardsLen = context->cardsLen;

    ShortDatabase* shortDatabase = chainDatabaseGpu->shortDatabase;
    LongDatabase* longDatabase = chainDatabaseGpu->longDatabase;

    int* longIndexes = chainDatabaseGpu->longIndexes;
    int longIndexesLen = chainDatabaseGpu->longIndexesLen;

    Chain** database = chainDatabaseGpu->database;
    int databaseLen = chainDatabaseGpu->databaseLen;
    
    //**************************************************************************
    // FILTER INDEXES
    
    int* indexesNew = NULL;
    int indexesNewLen;
    
    filterIndexesArray(&indexesNew, &indexesNewLen, indexes, indexesLen, 
        0, databaseLen - 1);
    
    int* longIndexesNew;
    int longIndexesNewLen;

    filterLongIndexesArray(&longIndexesNew, &longIndexesNewLen, longIndexes,
        longIndexesLen, indexesNew, indexesNewLen, databaseLen - 1);

    //**************************************************************************

    //**************************************************************************
    // INIT RESULTS
    
    *scores = (int*) malloc(queriesLen * databaseLen * sizeof(int));

    for (int i = 0; i < queriesLen; ++i) {
        for (int j = 0; j < databaseLen; ++j) {
            (*scores)[i * databaseLen + j] = NO_SCORE;
        }
    }

    //**************************************************************************
    
    //**************************************************************************
    // PREPARE CPU

    Mutex mutex;
    mutexCreate(&mutex);

    ContextCpu contextCpu;
    contextCpu.scores = *scores;
    contextCpu.type = type;
    contextCpu.queries = queries; 
    contextCpu.queriesLen = queriesLen;
    contextCpu.database = database;
    contextCpu.databaseLen = databaseLen;
    contextCpu.scorer = scorer;
    contextCpu.indexes = longIndexesNew;
    contextCpu.indexesLen = longIndexesNewLen;
    contextCpu.mutex = &mutex;
    contextCpu.lastIndexSolved = 0;
    contextCpu.cancelled = 0;

    //**************************************************************************

    //**************************************************************************
    // SOLVE MULTICARDED

    TIMER_START("Database solving GPU");

    Thread thread;
    threadCreate(&thread, scoreCpu, (void*) &contextCpu);

    TIMER_START("Short solve");
    
    scoreShortDatabasesGpu(*scores, type, queries, queriesLen, 
        shortDatabase, scorer, indexesNew, indexesNewLen, cards, cardsLen, NULL);

    TIMER_STOP;

    mutexLock(contextCpu.mutex);

    int longInexesSolved = contextCpu.lastIndexSolved;
    contextCpu.cancelled = 1;

    mutexUnlock(contextCpu.mutex);

    LOG("Long indexes solved CPU: \n%d\n\n", longInexesSolved);

    TIMER_START("Long solve");
    
    if (longInexesSolved < longIndexesNewLen) {
        scoreLongDatabasesGpu(*scores, type, queries, queriesLen,
            longDatabase, scorer, longIndexesNew + longInexesSolved, 
            longIndexesNewLen - longInexesSolved, cards, cardsLen, NULL);
    }

    TIMER_STOP;

    threadJoin(thread);

    TIMER_STOP;

    //**************************************************************************

    //**************************************************************************
    // CLEAN MEMORY
    
    mutexDelete(&mutex);

    if (longIndexesNew != longIndexes) {
        free(longIndexesNew);
    }

    free(indexesNew);
    free(param);
    
    //**************************************************************************
    
    return NULL;
}
//------------------------------------------------------------------------------

//------------------------------------------------------------------------------
// CPU WORKER

static void* scoreCpu(void* param) {

    ContextCpu* context = (ContextCpu*) param;

    int* scores = context->scores;
    int type = context->type;
    Chain** queries = context->queries; 
    int queriesLen = context->queriesLen;
    Chain** database_ = context->database;
    int databaseLen = context->databaseLen;
    Scorer* scorer = context->scorer;
    int* indexes = context->indexes;
    int indexesLen = context->indexesLen;
    Mutex* mutex = context->mutex;

    if (indexesLen == 0) {
        return NULL;
    }

    TIMER_START("Long indexes CPU: %d", indexesLen);

    //**************************************************************************
    // CREATE DATABASE
    
    Chain** database = (Chain**) malloc(indexesLen * sizeof(Chain*));

    for (int i = 0; i < indexesLen; ++i) {
        database[i] = database_[indexes[i]];
    }

    //**************************************************************************

    //**************************************************************************
    // PREPARE WORKER CONTEXT

    int* scoresCpu = (int*) malloc(queriesLen * indexesLen * sizeof(int));
    int lastQuery = 0;

    ContextWorkerCpu workerContext;
    workerContext.scores = scoresCpu;
    workerContext.type = type;
    workerContext.queries = queries; 
    workerContext.queriesLen = queriesLen;
    workerContext.database = database;
    workerContext.databaseLen = indexesLen;
    workerContext.scorer = scorer;
    workerContext.mutex = mutex;
    workerContext.lastQuery = &lastQuery;
    workerContext.lastTarget = &(context->lastIndexSolved);
    workerContext.cancelled = &(context->cancelled);

    //**************************************************************************

    //**************************************************************************
    // SOLVE MULTITHREADED

    int tasksNmr = CPU_THREADPOOL_STEP;
    ThreadPoolTask** tasks = (ThreadPoolTask**) malloc(tasksNmr * sizeof(ThreadPoolTask*));

    int over = 0;
    while (!over) {

        for (int i = 0; i < tasksNmr; ++i) {
            tasks[i] = threadPoolSubmitToFront(scoreCpuWorker, &workerContext);
        }
        
        for (int i = 0; i < tasksNmr; ++i) {
            threadPoolTaskWait(tasks[i]);
            threadPoolTaskDelete(tasks[i]);
        }

        mutexLock(mutex);

        if (context->cancelled || context->lastIndexSolved >= indexesLen) {
            over = 1;
        }

        mutexUnlock(mutex);
    }

    //**************************************************************************

    //**************************************************************************
    // SAVE SCORES

    int lastIndexSolved = context->lastIndexSolved;

    for (int i = 0; i < queriesLen; ++i) {
        for (int j = 0; j < lastIndexSolved; ++j) {
            scores[i * databaseLen + indexes[j]] = scoresCpu[i * indexesLen + j];
        }
    }

    //**************************************************************************

    //**************************************************************************
    // CLEAN MEMORY

    free(database);
    free(tasks);
    free(scoresCpu);

    //**************************************************************************

    TIMER_STOP;

    return NULL;
}

static void* scoreCpuWorker(void* param) {

    ContextWorkerCpu* context = (ContextWorkerCpu*) param;

    int* scores_ = context->scores;
    int type = context->type;
    Chain** queries = context->queries; 
    int queriesLen = context->queriesLen;
    Chain** database_ = context->database;
    int databaseLen = context->databaseLen;
    Scorer* scorer = context->scorer;
    Mutex* mutex = context->mutex;
    int* lastQuery = context->lastQuery;
    int* lastTarget = context->lastTarget;
    int* cancelled = context->cancelled;

    mutexLock(mutex);

    if (*lastQuery >= queriesLen) {
        *lastQuery = 0;
        *lastTarget += min(CPU_WORKER_STEP, databaseLen - *lastTarget);
    }

    int queryIdx = *lastQuery;
    int start = *lastTarget;
    int length = min(CPU_WORKER_STEP, databaseLen - start);

    if (start >= databaseLen || *cancelled) {
        mutexUnlock(mutex);
        return NULL;
    }

    (*lastQuery)++;

    mutexUnlock(mutex);

    int* scores = scores_ + queryIdx * databaseLen + start;

    Chain* query = queries[queryIdx];
    Chain** database = database_ + start;

    scoreDatabaseCpu(scores, type, query, database, length, scorer);

    return NULL;
}

//------------------------------------------------------------------------------

//------------------------------------------------------------------------------
// UTILS

static void filterIndexesArray(int** indexesNew, int* indexesNewLen, 
    int* indexes, int indexesLen, int minIndex, int maxIndex) {
    
    if (indexes == NULL) {
        *indexesNew = NULL;
        *indexesNewLen = 0;
        return;
    }
    
    *indexesNew = (int*) malloc(indexesLen * sizeof(int));
    *indexesNewLen = 0;

    for (int i = 0; i < indexesLen; ++i) {
    
        int idx = indexes[i];
        
        if (idx >= minIndex && idx <= maxIndex) {
            (*indexesNew)[*indexesNewLen] = idx;
            (*indexesNewLen)++;
        }
    }
}

static void filterLongIndexesArray(int** longIndexesNew, int* longIndexesNewLen, 
    int* longIndexes, int longIndexesLen, int* indexes, int indexesLen,
    int maxIndex) {

    if (indexes == NULL) {
        *longIndexesNew = longIndexes;
        *longIndexesNewLen = longIndexesLen;
        return;
    }

    int* mask = (int*) calloc(maxIndex + 1, sizeof(int));

    for (int i = 0; i < indexesLen; ++i) {
        mask[indexes[i]] = 1;
    }

    *longIndexesNew = (int*) malloc(longIndexesLen * sizeof(int));
    *longIndexesNewLen = 0;

    for (int i = 0; i < longIndexesLen; ++i) {
        if (mask[longIndexes[i]]) {
            (*longIndexesNew)[*longIndexesNewLen] = longIndexes[i];
            (*longIndexesNewLen)++;
        }
    }

    free(mask);
}

static int int2CmpY(const void* a_, const void* b_) {

    int2 a = *((int2*) a_);
    int2 b = *((int2*) b_);
    
    return a.y - b.y;
}

//------------------------------------------------------------------------------
//******************************************************************************

