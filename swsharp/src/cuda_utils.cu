/*
swsharp - CUDA parallelized Smith Waterman with applying Hirschberg's and 
Ukkonen's algorithm and dynamic cell pruning.
Copyright (C) 2013 Matija Korpar, contributor Mile Šikić

This program is free software: you can redistribute it and/or modify
it under the terms of the GNU General Public License as published by
the Free Software Foundation, either version 3 of the License, or
(at your option) any later version.

This program is distributed in the hope that it will be useful,
but WITHOUT ANY WARRANTY; without even the implied warranty of
MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
GNU General Public License for more details.

You should have received a copy of the GNU General Public License
along with this program.  If not, see <http://www.gnu.org/licenses/>.

Contact the author by mkorpar@gmail.com.
*/

#include <stdio.h>

#include "error.h"

#include "cuda_utils.h"

extern void cudaGetCards(int** cards, int* cardsLen) {
    
    hipGetDeviceCount(cardsLen);
    
    *cards = (int*) malloc(*cardsLen * sizeof(int));
    
    for (int i = 0; i < *cardsLen; ++i) {
        (*cards)[i] = i;   
    }
}

extern int cudaCheckCards(int* cards, int cardsLen) {
    
    int maxDeviceId;
    hipGetDeviceCount(&maxDeviceId);
    
    for (int i = 0; i < cardsLen; ++i) {
        if (cards[i] >= maxDeviceId) {
            return 0;
        }   
    }
    
    return 1;
}

extern void cudaCardBuckets(int*** cardBuckets, int** cardBucketsLens, 
    int* cards, int cardsLen, int buckets) {
    
    ASSERT(buckets <= cardsLen && buckets >= 1, "invalid bucket data");
    
    *cardBuckets = (int**) malloc(buckets * sizeof(int*));
    *cardBucketsLens = (int*) malloc(buckets * sizeof(int));
    
    memset(*cardBucketsLens, 0, buckets * sizeof(int));
    
    int i;
    
    int cardsLeft = cardsLen;
    i = 0;
    while (cardsLeft > 0) {
        (*cardBucketsLens)[i]++;
        i = (i + 1) % buckets;
        cardsLeft--;
    }
    
    int offset = 0;
    for (i = 0; i < buckets; ++i) {
       (*cardBuckets)[i] = cards + offset;
       offset += (*cardBucketsLens)[i];
    }
}
