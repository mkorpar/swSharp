#include "hip/hip_runtime.h"
/*
swsharp - CUDA parallelized Smith Waterman with applying Hirschberg's and 
Ukkonen's algorithm and dynamic cell pruning.
Copyright (C) 2013 Matija Korpar, contributor Mile Šikić

This program is free software: you can redistribute it and/or modify
it under the terms of the GNU General Public License as published by
the Free Software Foundation, either version 3 of the License, or
(at your option) any later version.

This program is distributed in the hope that it will be useful,
but WITHOUT ANY WARRANTY; without even the implied warranty of
MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
GNU General Public License for more details.

You should have received a copy of the GNU General Public License
along with this program.  If not, see <http://www.gnu.org/licenses/>.

Contact the author by mkorpar@gmail.com.
*/

#include <stdio.h>
#include <stdlib.h>
#include <string.h>

#include "chain.h"
#include "constants.h"
#include "cuda_utils.h"
#include "error.h"
#include "scorer.h"
#include "thread.h"
#include "utils.h"

#include "score_database_gpu_short.h"

#define THREADS   128
#define BLOCKS    120

#define INT4_ZERO make_int4(0, 0, 0, 0)
#define INT4_SCORE_MIN make_int4(SCORE_MIN, SCORE_MIN, SCORE_MIN, SCORE_MIN)

struct ShortDatabase {
    int length;
    int* order;
    int* positions;
    int blocks;
    int* offsets;
    int* lengths;
    int* lengthsPadded;
    size_t lengthsSize;
    char4* sequences;
    int sequencesLen;
    int sequencesRows;
    int sequencesCols;
    size_t sequencesSize;
};

typedef struct ShortDatabaseGpu {
    int* scores;
    int* lengths;
    int* lengthsPadded;
    hipArray* sequences;
    int2* hBus;
} ShortDatabaseGpu;

typedef struct Context {
    int** scores; 
    int type;
    Chain** queries;
    int queriesLen;
    ShortDatabase* shortDatabase;
    Scorer* scorer;
    int* indexes;
    int indexesLen;
    int* cards;
    int cardsLen;
} Context;

typedef struct KernelContext {
    int* scores; 
    int type;
    Chain** queries;
    int queriesLen;
    ShortDatabase* shortDatabase;
    Scorer* scorer;
    int* indexes;
    int indexesLen;
    int card;
    int queriesStart;
    int queriesStep;
    int blocksStart;
    int blocksStep;
} KernelContext;

static __constant__ int gapOpen_;
static __constant__ int gapExtend_;

static __constant__ int rows_;
static __constant__ int rowsPadded_;
static __constant__ int width_;

texture<int, 2, hipReadModeElementType> seqsTexture;
texture<char4, 2, hipReadModeElementType> subTexture;

//******************************************************************************
// PUBLIC

extern ShortDatabase* shortDatabaseCreate(Chain** database, int databaseLen);

extern void shortDatabaseDelete(ShortDatabase* shortDatabase);

extern void scoreShortDatabaseGpu(int** scores, int type, Chain* query, 
    ShortDatabase* shortDatabase, Scorer* scorer, int* indexes, int indexesLen, 
    int* cards, int cardsLen, Thread* thread);

extern void scoreShortDatabasesGpu(int** scores, int type, Chain** queries, 
    int queriesLen, ShortDatabase* shortDatabase, Scorer* scorer, int* indexes, 
    int indexesLen, int* cards, int cardsLen, Thread* thread);

//******************************************************************************

//******************************************************************************
// PRIVATE

static void scoreDatabase(int** scores, int type, Chain** queries, 
    int queriesLen, ShortDatabase* shortDatabase, Scorer* scorer, int* indexes, 
    int indexesLen, int* cards, int cardsLen, Thread* thread);

static void* scoreDatabaseThread(void* param);

// cpu kernels
static void* kernel(void* param);

static void kernelSingle(int* scores, int type, Chain* query,
    ShortDatabase* shortDatabase, ShortDatabaseGpu* shortDatabaseGpu, 
    Scorer* scorer, int blocksStart, int blocksStep);

// gpu database preparation
static ShortDatabaseGpu* shortDatabaseGpuCreate(ShortDatabase* shortDatabase,
    int* indexes, int indexesLen);

static void shortDatabaseGpuDelete(ShortDatabaseGpu* shortDatabaseGpu);

// gpu kernels
__device__ static int gap(int index);

__global__ static void hwSolveShortGpu(int* scores, int2* hBus, int* lengths, 
    int* lengthsPadded, int off);

__global__ static void nwSolveShortGpu(int* scores, int2* hBus, int* lengths, 
    int* lengthsPadded, int off);

__global__ static void swSolveShortGpu(int* scores, int2* hBus, int* lengths, 
    int* lengthsPadded, int off);

// utils
static int* createOrderArray(Chain** database, int databaseLen);

static int int2CmpY(const void* a_, const void* b_);

//******************************************************************************

//******************************************************************************
// PUBLIC

//------------------------------------------------------------------------------
// CONSTRUCTOR, DESTRUCTOR

extern ShortDatabase* shortDatabaseCreate(Chain** database, int databaseLen) {
    
    int* order = createOrderArray(database, databaseLen);
    
    int* positions = (int*) malloc(databaseLen * sizeof(int));
    
    for (int i = 0; i < databaseLen; ++i) {
        positions[order[i]] = i;
    }
    
    // create gpu database
    int sequencesCols = THREADS * BLOCKS;
    int sequencesRows = 0;

    int blocks = 0;
    
    // calculate sequence grid
    for (int i = sequencesCols - 1; i < databaseLen; i += sequencesCols) {
        int n = chainGetLength(database[order[i]]);
        sequencesRows += (n + (4 - n % 4) % 4) / 4;
        blocks++;
    }
    
    if (databaseLen % sequencesCols != 0) {
        int n = chainGetLength(database[order[databaseLen - 1]]);
        sequencesRows += (n + (4 - n % 4) % 4) / 4;
        blocks++;
    }
    
    // initialize structures
    int* offsets = (int*) malloc(blocks * sizeof(int));
    offsets[0] = 0;
    
    size_t lengthsSize = blocks * sequencesCols * sizeof(int);
    int* lengths = (int*) malloc(lengthsSize);
    memset(lengths, 0, lengthsSize);
    
    int* lengthsPadded = (int*) malloc(lengthsSize);
    memset(lengthsPadded, 0, lengthsSize);
    
    size_t sequencesSize = sequencesRows * sequencesCols * sizeof(char4);
    char4* sequences = (char4*) malloc(sequencesSize);
    
    // tmp
    size_t sequenceSize = chainGetLength(database[order[databaseLen - 1]]) + 4;
    char* sequence = (char*) malloc(sequenceSize);

    for(int i = 0, j = 0, cx = 0, cy = 0; i < databaseLen; i++){

        //get the sequence and its length
        Chain* chain = database[order[i]];
        int n = chainGetLength(chain);    
        
        lengths[j * sequencesCols + cx] = n;
        
        chainCopyCodes(chain, sequence);
        memset(sequence + n, 255, 4 * sizeof(char));

        n = n + (4 - n % 4) % 4;
        int n4 = n / 4;

        lengthsPadded[j * sequencesCols + cx] = n4;
        
        char4* ptr = sequences + cy * sequencesCols + cx;
        for(int k = 0; k < n; k += 4){
            ptr->x = sequence[k];
            ptr->y = sequence[k + 1];
            ptr->z = sequence[k + 2];
            ptr->w = sequence[k + 3];
            ptr += sequencesCols;
        }

        cx++;
        
        if(cx == sequencesCols){
            offsets[j + 1] = offsets[j] + n4;
            cx = 0;
            cy += n4;
            j++;
        }
    }
    
    free(sequence);
    
    ShortDatabase* shortDatabase = 
        (ShortDatabase*) malloc(sizeof(struct ShortDatabase));
    
    shortDatabase->length = databaseLen;
    shortDatabase->order = order;
    shortDatabase->positions = positions;
    shortDatabase->blocks = blocks;
    shortDatabase->offsets = offsets;
    shortDatabase->lengths = lengths;
    shortDatabase->lengthsPadded = lengthsPadded;
    shortDatabase->lengthsSize = lengthsSize;
    shortDatabase->sequences = sequences;
    shortDatabase->sequencesSize = sequencesSize;
    shortDatabase->sequencesLen = databaseLen;
    shortDatabase->sequencesRows = sequencesRows;
    shortDatabase->sequencesCols = sequencesCols;
    
    return shortDatabase;
}
    
extern void shortDatabaseDelete(ShortDatabase* shortDatabase) {

    free(shortDatabase->order);
    free(shortDatabase->positions);
    free(shortDatabase->offsets);
    free(shortDatabase->lengths);
    free(shortDatabase->lengthsPadded);
    free(shortDatabase->sequences);
    
    free(shortDatabase);
    shortDatabase = NULL;
}

//------------------------------------------------------------------------------

//------------------------------------------------------------------------------
// DATABASE SCORING

extern void scoreShortDatabaseGpu(int** scores, int type, Chain* query, 
    ShortDatabase* shortDatabase, Scorer* scorer, int* indexes, int indexesLen, 
    int* cards, int cardsLen, Thread* thread) {
    scoreDatabase(scores, type, &query, 1, shortDatabase, scorer, indexes, 
        indexesLen, cards, cardsLen, thread);
}

extern void scoreShortDatabasesGpu(int** scores, int type, Chain** queries, 
    int queriesLen, ShortDatabase* shortDatabase, Scorer* scorer, int* indexes, 
    int indexesLen, int* cards, int cardsLen, Thread* thread) {
    scoreDatabase(scores, type, queries, queriesLen, shortDatabase, scorer,
        indexes, indexesLen, cards, cardsLen, thread);
}

//------------------------------------------------------------------------------

//******************************************************************************

//******************************************************************************
// PRIVATE

//------------------------------------------------------------------------------
// DATABASE SCORING

static void scoreDatabase(int** scores, int type, Chain** queries, 
    int queriesLen, ShortDatabase* shortDatabase, Scorer* scorer, int* indexes, 
    int indexesLen, int* cards, int cardsLen, Thread* thread) {
    
    ASSERT(cardsLen > 0, "no GPUs available");
    
    Context* param = (Context*) malloc(sizeof(Context));
    
    param->scores = scores;
    param->type = type;
    param->queries = queries;
    param->queriesLen = queriesLen;
    param->shortDatabase = shortDatabase;
    param->scorer = scorer;
    param->indexes = indexes;
    param->indexesLen = indexesLen;
    param->cards = cards;
    param->cardsLen = cardsLen;
    
    if (thread == NULL) {
        scoreDatabaseThread(param);
    } else {
        threadCreate(thread, scoreDatabaseThread, (void*) param);
    }
}

static void* scoreDatabaseThread(void* param) {

    Context* context = (Context*) param;
    
    int** scores = context->scores;
    int type = context->type;
    Chain** queries = context->queries;
    int queriesLen = context->queriesLen;
    ShortDatabase* shortDatabase = context->shortDatabase;
    Scorer* scorer = context->scorer;
    int* indexes = context->indexes;
    int indexesLen = context->indexesLen;
    int* cards = context->cards;
    int cardsLen = context->cardsLen;
    
    int databaseLen = shortDatabase->length;
    int* order = shortDatabase->order;
    
    //**************************************************************************
    // SOLVE MULTICARDED
    
    // there is no level of paralelization available
    if (queriesLen == 1 && shortDatabase->blocks == 1) {
        cardsLen = 1;
    }
    
    int threadNmr = cardsLen;
    
    Thread* threads = (Thread*) malloc((threadNmr - 1) * sizeof(Thread));
    
    int* unordered = (int*) malloc(queriesLen * databaseLen * sizeof(int));
    
    KernelContext* contexts = 
        (KernelContext*) malloc(threadNmr * sizeof(KernelContext));

    for (int i = 0; i < threadNmr; ++i) {
    
        contexts[i].scores = unordered;
        contexts[i].type = type;
        contexts[i].queries = queries;
        contexts[i].queriesLen = queriesLen;
        contexts[i].shortDatabase = shortDatabase;
        contexts[i].scorer = scorer;
        contexts[i].card = cards[i];
        contexts[i].indexes = indexes;
        contexts[i].indexesLen = indexesLen;
        
        if (threadNmr < queriesLen) {
            // one query, single card
            contexts[i].queriesStart = i;
            contexts[i].queriesStep = cardsLen;
            contexts[i].blocksStart = 0;
            contexts[i].blocksStep = 1;
        } else {
            // one query, multiple cards
            contexts[i].queriesStart = 0;
            contexts[i].queriesStep = 1;
            contexts[i].blocksStart = i;
            contexts[i].blocksStep = cardsLen;
        }
    }

    for (int i = 0; i < threadNmr - 1; ++i) {    
        threadCreate(&threads[i], kernel, &contexts[i]);
    }
    
    kernel(&contexts[threadNmr - 1]);
    
    for (int i = 0; i < threadNmr - 1; ++i) {
        threadJoin(threads[i]);
    }
    
    //**************************************************************************
    
    //**************************************************************************
    // SAVE SCORES
    
    *scores = (int*) malloc(queriesLen * databaseLen * sizeof(int));
    
    // copy
    for (int i = 0; i < queriesLen; ++i) {
        for (int j = 0; j < databaseLen; ++j) {
            (*scores)[i * databaseLen + order[j]] = unordered[i * databaseLen + j];
        }
    }
    
    //**************************************************************************
    
    //**************************************************************************
    // CLEAN MEMORY

    free(unordered);
    free(threads);
    free(contexts);

    free(param);
    
    //**************************************************************************
    
    return NULL;
}

//------------------------------------------------------------------------------

//------------------------------------------------------------------------------
// CPU KERNELS

static void* kernel(void* param) {

    KernelContext* context = (KernelContext*) param;
    
    int* scores = context->scores;
    int type = context->type;
    Chain** queries = context->queries;
    int queriesLen = context->queriesLen;
    ShortDatabase* shortDatabase = context->shortDatabase;
    Scorer* scorer = context->scorer;
    int* indexes = context->indexes;
    int indexesLen = context->indexesLen;
    int card = context->card;
    int queriesStart = context->queriesStart;
    int queriesStep = context->queriesStep;
    int blocksStart = context->blocksStart;
    int blocksStep = context->blocksStep;
    
    // set card
    int currentCard;
    CUDA_SAFE_CALL(hipGetDevice(&currentCard));
    if (currentCard != card) {
        CUDA_SAFE_CALL(hipDeviceReset());
        CUDA_SAFE_CALL(hipSetDevice(card));
    }
    
    // prepare gpu db
    ShortDatabaseGpu* shortDatabaseGpu = shortDatabaseGpuCreate(shortDatabase, 
        indexes, indexesLen);

    // solve
    for (int i = queriesStart; i < queriesLen; i += queriesStep) {
    
        Chain* query = queries[i];
        int offset = i * shortDatabase->length;
        
        kernelSingle(scores + offset, type, query, shortDatabase, 
            shortDatabaseGpu, scorer, blocksStart, blocksStep);
    }
    
    shortDatabaseGpuDelete(shortDatabaseGpu);

    return NULL;
}

static void kernelSingle(int* scores, int type, Chain* query,
    ShortDatabase* shortDatabase, ShortDatabaseGpu* shortDatabaseGpu, 
    Scorer* scorer, int blocksStart, int blocksStep) {
    
    int gapOpen = scorerGetGapOpen(scorer);
    int gapExtend = scorerGetGapExtend(scorer);
    
    int* offsets = shortDatabase->offsets;
    int blocks = shortDatabase->blocks;
    
    int sequencesCols = shortDatabase->sequencesCols;
    int sequencesLen = shortDatabase->sequencesLen;
    
    int* scoresGpu = shortDatabaseGpu->scores;
    int* lengths = shortDatabaseGpu->lengths;
    int* lengthsPadded = shortDatabaseGpu->lengthsPadded;
    
    int2* hBus = shortDatabaseGpu->hBus;
    
    //**************************************************************************
    // CREATE QUERY PROFILE
    
    int rows = chainGetLength(query);
    int rowsGpu = rows + (8 - rows % 8) % 8;
    
    size_t rowSize = rows * sizeof(char);
    char* row = (char*) malloc(rowSize);
    chainCopyCodes(query, row);

    int subLen = scorerGetMaxCode(scorer) + 1;
    size_t subSize = rowsGpu * subLen * sizeof(char);
    char4* subCpu = (char4*) malloc(subSize);
    memset(subCpu, 0, subSize);
    for (int i = 0; i < rowsGpu / 4; ++i) {
        for (int j = 0; j < subLen - 1; ++j) {
            char4 scr;
            scr.x = i * 4 + 0 >= rows ? 0 : scorerScore(scorer, row[i * 4 + 0], j);
            scr.y = i * 4 + 1 >= rows ? 0 : scorerScore(scorer, row[i * 4 + 1], j);
            scr.z = i * 4 + 2 >= rows ? 0 : scorerScore(scorer, row[i * 4 + 2], j);
            scr.w = i * 4 + 3 >= rows ? 0 : scorerScore(scorer, row[i * 4 + 3], j);
            subCpu[i * subLen + j] = scr;
        }
    }
    
    hipArray* subArray;
    int subH = rowsGpu / 4;
    CUDA_SAFE_CALL(hipMallocArray(&subArray, &subTexture.channelDesc, subLen, subH)); 
    CUDA_SAFE_CALL(hipMemcpyToArray (subArray, 0, 0, subCpu, subSize, TO_GPU));
    CUDA_SAFE_CALL(hipBindTextureToArray(subTexture, subArray));
    subTexture.addressMode[0] = hipAddressModeClamp;
    subTexture.addressMode[1] = hipAddressModeClamp;
    subTexture.filterMode = hipFilterModePoint;
    subTexture.normalized = false;

    //**************************************************************************
    
    //**************************************************************************
    // INIT GPU
    
    CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(rows_), &rows, sizeof(int)));
    CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(rowsPadded_), &rowsGpu, sizeof(int)));
    CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(gapOpen_), &gapOpen, sizeof(int)));
    CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(gapExtend_), &gapExtend, sizeof(int)));
    
    //**************************************************************************
    
    //**************************************************************************
    // SOVLE
    
    void (*function)(int*, int2*, int*, int*, int);
    switch (type) {
    case SW_ALIGN: 
        function = swSolveShortGpu;
        break;
    case NW_ALIGN: 
        function = nwSolveShortGpu;
        break;
    case HW_ALIGN:
        function = hwSolveShortGpu;
        break;
    default:
        ERROR("Wrong align type");
    }
    
    for (int i = blocksStart; i < blocks; i += blocksStep) {

        int colOff = i * sequencesCols;
        int rowOff = offsets[i];
        
        function<<<BLOCKS, THREADS>>>(scoresGpu, hBus, lengths + colOff, 
            lengthsPadded + colOff, rowOff);

        // copy scores from the GPU
        size_t size = min(sequencesCols, sequencesLen - colOff) * sizeof(int);
        CUDA_SAFE_CALL(hipMemcpy(scores + colOff, scoresGpu, size, FROM_GPU));
    }
    
    //**************************************************************************
    
    //**************************************************************************
    // CLEAN MEMORY
    
    free(subCpu);
    free(row);
    
    CUDA_SAFE_CALL(hipFreeArray(subArray));
    
    //**************************************************************************
}

//------------------------------------------------------------------------------

//------------------------------------------------------------------------------
// GPU DATABASE PREPARE

extern ShortDatabaseGpu* shortDatabaseGpuCreate(ShortDatabase* shortDatabase,
    int* indexes, int indexesLen) {
    
    int* lengthsPadded;
    
    // filter and copy padded lengths
    if (indexes == NULL) {
        lengthsPadded = shortDatabase->lengthsPadded;
    } else {
        lengthsPadded = (int*) malloc(shortDatabase->lengthsSize);
        memset(lengthsPadded, 0, shortDatabase->lengthsSize);
        
        int length = shortDatabase->length;
        
        for (int i = 0; i < indexesLen; ++i) {
        
            int index = indexes[i];
            ASSERT(index < length, "wrong index: %d\n", index);
            
            int ord = shortDatabase->positions[index];
            lengthsPadded[ord] = shortDatabase->lengthsPadded[ord];
        }
    }
    
    size_t lengthsSize = shortDatabase->lengthsSize;
    int* lengthsPaddedGpu;
    CUDA_SAFE_CALL(hipMalloc(&lengthsPaddedGpu, lengthsSize));
    CUDA_SAFE_CALL(hipMemcpy(lengthsPaddedGpu, lengthsPadded, lengthsSize, TO_GPU));
    
    if (indexes != NULL) {
        free(lengthsPadded);
    }

    // copy lengths
    int* lengths = shortDatabase->lengths;
    int* lengthsGpu;
    CUDA_SAFE_CALL(hipMalloc(&lengthsGpu, lengthsSize));
    CUDA_SAFE_CALL(hipMemcpy(lengthsGpu, lengths, lengthsSize, TO_GPU));
    
    // copy sequences
    int sequencesCols = shortDatabase->sequencesCols;
    int sequencesRows = shortDatabase->sequencesRows;
    size_t sequencesSize = shortDatabase->sequencesSize;
    char4* sequences = shortDatabase->sequences;
    hipArray* sequencesGpu;
    hipChannelFormatDesc channel = seqsTexture.channelDesc;
    CUDA_SAFE_CALL(hipMallocArray(&sequencesGpu, &channel, sequencesCols, sequencesRows)); 
    CUDA_SAFE_CALL(hipMemcpyToArray(sequencesGpu, 0, 0, sequences, sequencesSize, TO_GPU));
    CUDA_SAFE_CALL(hipBindTextureToArray(seqsTexture, sequencesGpu));
    
    // init scores
    int* scoresGpu;
    size_t scoresSize = sequencesCols * sizeof(int);
    CUDA_SAFE_CALL(hipMalloc(&scoresGpu, scoresSize));
    
    // init h bus
    int* offsets = shortDatabase->offsets;
    int blocks = shortDatabase->blocks;
    int2* hBusGpu;
    int hBusHeight = (sequencesRows - offsets[blocks - 1]) * 4;
    size_t hBusSize = sequencesCols * hBusHeight * sizeof(int2);
    CUDA_SAFE_CALL(hipMalloc(&hBusGpu, hBusSize));
    
    // constants
    CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(width_), &sequencesCols, sizeof(int)));
    
    ShortDatabaseGpu* shortDatabaseGpu = 
        (ShortDatabaseGpu*) malloc(sizeof(struct ShortDatabaseGpu));
    
    shortDatabaseGpu->scores = scoresGpu;
    shortDatabaseGpu->lengths = lengthsGpu;
    shortDatabaseGpu->lengthsPadded = lengthsPaddedGpu;
    shortDatabaseGpu->sequences = sequencesGpu;
    shortDatabaseGpu->hBus = hBusGpu;

    return shortDatabaseGpu;
}

extern void shortDatabaseGpuDelete(ShortDatabaseGpu* shortDatabaseGpu) {

    CUDA_SAFE_CALL(hipFree(shortDatabaseGpu->scores));
    CUDA_SAFE_CALL(hipFree(shortDatabaseGpu->lengths));
    CUDA_SAFE_CALL(hipFree(shortDatabaseGpu->lengthsPadded));
    CUDA_SAFE_CALL(hipFreeArray(shortDatabaseGpu->sequences));
    CUDA_SAFE_CALL(hipFree(shortDatabaseGpu->hBus));
    
    CUDA_SAFE_CALL(hipUnbindTexture(seqsTexture));
    
    free(shortDatabaseGpu);
    shortDatabaseGpu = NULL;
}

//------------------------------------------------------------------------------

//------------------------------------------------------------------------------
// GPU KERNELS

__device__ static int gap(int index) {
    return (-gapOpen_ - index * gapExtend_) * (index >= 0);
}

__global__ static void hwSolveShortGpu(int* scores, int2* hBus, int* lengths, 
    int* lengthsPadded, int off) {

    int id = threadIdx.x + blockIdx.x * blockDim.x;
    int cols = lengthsPadded[id];

    if (cols == 0) {
        scores[id] = NO_SCORE;
        return;
    }
    
    int score = SCORE_MIN;
    
    int4 scrUp;
    int4 affUp;
    int4 mchUp;
    
    int4 scrDown;
    int4 affDown;
    int4 mchDown;
    
    int2 wBus;
    int del;
    
    int lastRow = rows_ - 1;
    int realCols = lengths[id];
    
    for (int j = 0; j < cols * 4; ++j) {
        hBus[j * width_ + id] = make_int2(0, SCORE_MIN);
    }
    
    for (int i = 0; i < rowsPadded_; i += 8) {
    
        scrUp = make_int4(gap(i), gap(i + 1), gap(i + 2), gap(i + 3));
        affUp = INT4_SCORE_MIN;
        mchUp = make_int4(gap(i - 1), gap(i), gap(i + 1), gap(i + 2));
        
        scrDown = make_int4(gap(i + 4), gap(i + 5), gap(i + 6), gap(i + 7));
        affDown = INT4_SCORE_MIN;
        mchDown = make_int4(gap(i + 3), gap(i + 4), gap(i + 5), gap(i + 6));
        
        for (int j = 0; j < cols; ++j) {
        
            int columnCodes = tex2D(seqsTexture, id, j + off);
            
            #pragma unroll
            for (int k = 0; k < 4; ++k) {
            
                int validCol = (j * 4 + k) < realCols;
                
                wBus = hBus[(j * 4 + k) * width_ + id];
                
                char code = (columnCodes >> (k << 3));
                char4 rowScores = tex2D(subTexture, code, i / 4);
                
                del = max(wBus.x - gapOpen_, wBus.y - gapExtend_);
                affUp.x = max(scrUp.x - gapOpen_, affUp.x - gapExtend_);
                scrUp.x = mchUp.x + rowScores.x; 
                scrUp.x = max(scrUp.x, del);
                scrUp.x = max(scrUp.x, affUp.x);
                mchUp.x = wBus.x;
                if (i + 0 == lastRow && validCol) score = max(score, scrUp.x);
                
                del = max(scrUp.x - gapOpen_, del - gapExtend_);
                affUp.y = max(scrUp.y - gapOpen_, affUp.y - gapExtend_);
                scrUp.y = mchUp.y + rowScores.y; 
                scrUp.y = max(scrUp.y, del);
                scrUp.y = max(scrUp.y, affUp.y);
                mchUp.y = scrUp.x;
                if (i + 1 == lastRow && validCol) score = max(score, scrUp.y);
                
                del = max(scrUp.y - gapOpen_, del - gapExtend_);
                affUp.z = max(scrUp.z - gapOpen_, affUp.z - gapExtend_);
                scrUp.z = mchUp.z + rowScores.z; 
                scrUp.z = max(scrUp.z, del);
                scrUp.z = max(scrUp.z, affUp.z);
                mchUp.z = scrUp.y;
                if (i + 2 == lastRow && validCol) score = max(score, scrUp.z);
                
                del = max(scrUp.z - gapOpen_, del - gapExtend_);
                affUp.w = max(scrUp.w - gapOpen_, affUp.w - gapExtend_);
                scrUp.w = mchUp.w + rowScores.w; 
                scrUp.w = max(scrUp.w, del);
                scrUp.w = max(scrUp.w, affUp.w);
                mchUp.w = scrUp.z;
                if (i + 3 == lastRow && validCol) score = max(score, scrUp.w);

                rowScores = tex2D(subTexture, code, i / 4 + 1);
                
                del = max(scrUp.w - gapOpen_, del - gapExtend_);
                affDown.x = max(scrDown.x - gapOpen_, affDown.x - gapExtend_);
                scrDown.x = mchDown.x + rowScores.x; 
                scrDown.x = max(scrDown.x, del);
                scrDown.x = max(scrDown.x, affDown.x);
                mchDown.x = scrUp.w;
                if (i + 4 == lastRow && validCol) score = max(score, scrDown.x);
                
                del = max(scrDown.x - gapOpen_, del - gapExtend_);
                affDown.y = max(scrDown.y - gapOpen_, affDown.y - gapExtend_);
                scrDown.y = mchDown.y + rowScores.y; 
                scrDown.y = max(scrDown.y, del);
                scrDown.y = max(scrDown.y, affDown.y);
                mchDown.y = scrDown.x;
                if (i + 5 == lastRow && validCol) score = max(score, scrDown.y);
                
                del = max(scrDown.y - gapOpen_, del - gapExtend_);
                affDown.z = max(scrDown.z - gapOpen_, affDown.z - gapExtend_);
                scrDown.z = mchDown.z + rowScores.z; 
                scrDown.z = max(scrDown.z, del);
                scrDown.z = max(scrDown.z, affDown.z);
                mchDown.z = scrDown.y;
                if (i + 6 == lastRow && validCol) score = max(score, scrDown.z);
                
                del = max(scrDown.z - gapOpen_, del - gapExtend_);
                affDown.w = max(scrDown.w - gapOpen_, affDown.w - gapExtend_);
                scrDown.w = mchDown.w + rowScores.w; 
                scrDown.w = max(scrDown.w, del);
                scrDown.w = max(scrDown.w, affDown.w);
                mchDown.w = scrDown.z;
                if (i + 7 == lastRow && validCol) score = max(score, scrDown.w);
                
                wBus.x = scrDown.w;
                wBus.y = del;
                
                hBus[(j * 4 + k) * width_ + id] = wBus;
            }
        }
    }
    
    scores[id] = score;
}

__global__ static void nwSolveShortGpu(int* scores, int2* hBus, int* lengths, 
    int* lengthsPadded, int off) {

    int id = threadIdx.x + blockIdx.x * blockDim.x;
    int cols = lengthsPadded[id];

    if (cols == 0) {
        scores[id] = NO_SCORE;
        return;
    }
    
    int score = SCORE_MIN;
    
    int4 scrUp;
    int4 affUp;
    int4 mchUp;
    
    int4 scrDown;
    int4 affDown;
    int4 mchDown;
    
    int2 wBus;
    int del;
    
    int lastRow = rows_ - 1;
    int realCols = lengths[id];
    
    for (int j = 0; j < cols * 4; ++j) {
        hBus[j * width_ + id] = make_int2(gap(j), SCORE_MIN);
    }
    
    for (int i = 0; i < rowsPadded_; i += 8) {
    
        scrUp = make_int4(gap(i), gap(i + 1), gap(i + 2), gap(i + 3));
        affUp = INT4_SCORE_MIN;
        mchUp = make_int4(gap(i - 1), gap(i), gap(i + 1), gap(i + 2));
        
        scrDown = make_int4(gap(i + 4), gap(i + 5), gap(i + 6), gap(i + 7));
        affDown = INT4_SCORE_MIN;
        mchDown = make_int4(gap(i + 3), gap(i + 4), gap(i + 5), gap(i + 6));
        
        for (int j = 0; j < cols; ++j) {
        
            int columnCodes = tex2D(seqsTexture, id, j + off);
            
            #pragma unroll
            for (int k = 0; k < 4; ++k) {
            
                int lastCol = (j * 4 + k) == (realCols - 1);
                
                wBus = hBus[(j * 4 + k) * width_ + id];
                
                char code = (columnCodes >> (k << 3));
                char4 rowScores = tex2D(subTexture, code, i / 4);
                
                del = max(wBus.x - gapOpen_, wBus.y - gapExtend_);
                affUp.x = max(scrUp.x - gapOpen_, affUp.x - gapExtend_);
                scrUp.x = mchUp.x + rowScores.x; 
                scrUp.x = max(scrUp.x, del);
                scrUp.x = max(scrUp.x, affUp.x);
                mchUp.x = wBus.x;
                if (i + 0 == lastRow && lastCol) score = scrUp.x;
                
                del = max(scrUp.x - gapOpen_, del - gapExtend_);
                affUp.y = max(scrUp.y - gapOpen_, affUp.y - gapExtend_);
                scrUp.y = mchUp.y + rowScores.y; 
                scrUp.y = max(scrUp.y, del);
                scrUp.y = max(scrUp.y, affUp.y);
                mchUp.y = scrUp.x;
                if (i + 1 == lastRow && lastCol) score = scrUp.y;
                
                del = max(scrUp.y - gapOpen_, del - gapExtend_);
                affUp.z = max(scrUp.z - gapOpen_, affUp.z - gapExtend_);
                scrUp.z = mchUp.z + rowScores.z; 
                scrUp.z = max(scrUp.z, del);
                scrUp.z = max(scrUp.z, affUp.z);
                mchUp.z = scrUp.y;
                if (i + 2 == lastRow && lastCol) score = scrUp.z;
                
                del = max(scrUp.z - gapOpen_, del - gapExtend_);
                affUp.w = max(scrUp.w - gapOpen_, affUp.w - gapExtend_);
                scrUp.w = mchUp.w + rowScores.w; 
                scrUp.w = max(scrUp.w, del);
                scrUp.w = max(scrUp.w, affUp.w);
                mchUp.w = scrUp.z;
                if (i + 3 == lastRow && lastCol) score = scrUp.w;

                rowScores = tex2D(subTexture, code, i / 4 + 1);
                
                del = max(scrUp.w - gapOpen_, del - gapExtend_);
                affDown.x = max(scrDown.x - gapOpen_, affDown.x - gapExtend_);
                scrDown.x = mchDown.x + rowScores.x; 
                scrDown.x = max(scrDown.x, del);
                scrDown.x = max(scrDown.x, affDown.x);
                mchDown.x = scrUp.w;
                if (i + 4 == lastRow && lastCol) score = scrDown.x;
                
                del = max(scrDown.x - gapOpen_, del - gapExtend_);
                affDown.y = max(scrDown.y - gapOpen_, affDown.y - gapExtend_);
                scrDown.y = mchDown.y + rowScores.y; 
                scrDown.y = max(scrDown.y, del);
                scrDown.y = max(scrDown.y, affDown.y);
                mchDown.y = scrDown.x;
                if (i + 5 == lastRow && lastCol) score = scrDown.y;
                
                del = max(scrDown.y - gapOpen_, del - gapExtend_);
                affDown.z = max(scrDown.z - gapOpen_, affDown.z - gapExtend_);
                scrDown.z = mchDown.z + rowScores.z; 
                scrDown.z = max(scrDown.z, del);
                scrDown.z = max(scrDown.z, affDown.z);
                mchDown.z = scrDown.y;
                if (i + 6 == lastRow && lastCol) score = scrDown.z;
                
                del = max(scrDown.z - gapOpen_, del - gapExtend_);
                affDown.w = max(scrDown.w - gapOpen_, affDown.w - gapExtend_);
                scrDown.w = mchDown.w + rowScores.w; 
                scrDown.w = max(scrDown.w, del);
                scrDown.w = max(scrDown.w, affDown.w);
                mchDown.w = scrDown.z;
                if (i + 7 == lastRow && lastCol) score = scrDown.w;
                
                wBus.x = scrDown.w;
                wBus.y = del;
                
                hBus[(j * 4 + k) * width_ + id] = wBus;
            }
        }
    }
    
    scores[id] = score;
}

__global__ static void swSolveShortGpu(int* scores, int2* hBus, int* lengths, 
    int* lengthsPadded, int off) {

    int id = threadIdx.x + blockIdx.x * blockDim.x;
    int cols = lengthsPadded[id];

    if (cols == 0) {
        scores[id] = NO_SCORE;
        return;
    }
    
    int score = 0;
    
    int4 scrUp;
    int4 affUp;
    int4 mchUp;
    
    int4 scrDown;
    int4 affDown;
    int4 mchDown;
    
    int2 wBus;
    int del;
    
    for (int j = 0; j < cols * 4; ++j) {
        hBus[j * width_ + id] = make_int2(0, SCORE_MIN);
    }
    
    for (int i = 0; i < rowsPadded_; i += 8) {
    
        scrUp = INT4_ZERO;
        affUp = INT4_SCORE_MIN;
        mchUp = INT4_ZERO;
        
        scrDown = INT4_ZERO;
        affDown = INT4_SCORE_MIN;
        mchDown = INT4_ZERO;
        
        for (int j = 0; j < cols; ++j) {
        
            int columnCodes = tex2D(seqsTexture, id, j + off);
            
            #pragma unroll
            for (int k = 0; k < 4; ++k) {
            
                wBus = hBus[(j * 4 + k) * width_ + id];
                
                char code = (columnCodes >> (k << 3));
                char4 rowScores = tex2D(subTexture, code, i / 4);
                
                del = max(wBus.x - gapOpen_, wBus.y - gapExtend_);
                affUp.x = max(scrUp.x - gapOpen_, affUp.x - gapExtend_);
                scrUp.x = mchUp.x + rowScores.x; 
                scrUp.x = max(scrUp.x, del);
                scrUp.x = max(scrUp.x, affUp.x);
                scrUp.x = max(scrUp.x, 0);
                mchUp.x = wBus.x;
                score = max(score, scrUp.x);
                
                del = max(scrUp.x - gapOpen_, del - gapExtend_);
                affUp.y = max(scrUp.y - gapOpen_, affUp.y - gapExtend_);
                scrUp.y = mchUp.y + rowScores.y; 
                scrUp.y = max(scrUp.y, del);
                scrUp.y = max(scrUp.y, affUp.y);
                scrUp.y = max(scrUp.y, 0);
                mchUp.y = scrUp.x;
                score = max(score, scrUp.y);
                
                del = max(scrUp.y - gapOpen_, del - gapExtend_);
                affUp.z = max(scrUp.z - gapOpen_, affUp.z - gapExtend_);
                scrUp.z = mchUp.z + rowScores.z; 
                scrUp.z = max(scrUp.z, del);
                scrUp.z = max(scrUp.z, affUp.z);
                scrUp.z = max(scrUp.z, 0);
                mchUp.z = scrUp.y;
                score = max(score, scrUp.z);
                
                del = max(scrUp.z - gapOpen_, del - gapExtend_);
                affUp.w = max(scrUp.w - gapOpen_, affUp.w - gapExtend_);
                scrUp.w = mchUp.w + rowScores.w; 
                scrUp.w = max(scrUp.w, del);
                scrUp.w = max(scrUp.w, affUp.w);
                scrUp.w = max(scrUp.w, 0);
                mchUp.w = scrUp.z;
                score = max(score, scrUp.w);

                rowScores = tex2D(subTexture, code, i / 4 + 1);
                
                del = max(scrUp.w - gapOpen_, del - gapExtend_);
                affDown.x = max(scrDown.x - gapOpen_, affDown.x - gapExtend_);
                scrDown.x = mchDown.x + rowScores.x; 
                scrDown.x = max(scrDown.x, del);
                scrDown.x = max(scrDown.x, affDown.x);
                scrDown.x = max(scrDown.x, 0);
                mchDown.x = scrUp.w;
                score = max(score, scrDown.x);
                
                del = max(scrDown.x - gapOpen_, del - gapExtend_);
                affDown.y = max(scrDown.y - gapOpen_, affDown.y - gapExtend_);
                scrDown.y = mchDown.y + rowScores.y; 
                scrDown.y = max(scrDown.y, del);
                scrDown.y = max(scrDown.y, affDown.y);
                scrDown.y = max(scrDown.y, 0);
                mchDown.y = scrDown.x;
                score = max(score, scrDown.y);
                
                del = max(scrDown.y - gapOpen_, del - gapExtend_);
                affDown.z = max(scrDown.z - gapOpen_, affDown.z - gapExtend_);
                scrDown.z = mchDown.z + rowScores.z; 
                scrDown.z = max(scrDown.z, del);
                scrDown.z = max(scrDown.z, affDown.z);
                scrDown.z = max(scrDown.z, 0);
                mchDown.z = scrDown.y;
                score = max(score, scrDown.z);
                
                del = max(scrDown.z - gapOpen_, del - gapExtend_);
                affDown.w = max(scrDown.w - gapOpen_, affDown.w - gapExtend_);
                scrDown.w = mchDown.w + rowScores.w; 
                scrDown.w = max(scrDown.w, del);
                scrDown.w = max(scrDown.w, affDown.w);
                scrDown.w = max(scrDown.w, 0);
                mchDown.w = scrDown.z;
                score = max(score, scrDown.w);
                
                wBus.x = scrDown.w;
                wBus.y = del;
                
                hBus[(j * 4 + k) * width_ + id] = wBus;
            }
        }
    }
    
    scores[id] = score;
}

//------------------------------------------------------------------------------

//------------------------------------------------------------------------------
// UTILS

static int* createOrderArray(Chain** database, int databaseLen) {

    int2* packed = (int2*) malloc(databaseLen * sizeof(int2));
    
    for (int i = 0; i < databaseLen; ++i) {
        packed[i].x = i;
        packed[i].y = chainGetLength(database[i]);
    }
    
    qsort(packed, databaseLen, sizeof(int2), int2CmpY);

    int* order = (int*) malloc(databaseLen * sizeof(int));
    
    for (int i = 0; i < databaseLen; ++i) {
        order[i] = packed[i].x;
    }
    
    free(packed);
    
    return order;
}

static int int2CmpY(const void* a_, const void* b_) {

    int2 a = *((int2*) a_);
    int2 b = *((int2*) b_);
    
    return a.y - b.y;
}

//------------------------------------------------------------------------------
//******************************************************************************
