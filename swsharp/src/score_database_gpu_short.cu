#include "hip/hip_runtime.h"
/*
swsharp - CUDA parallelized Smith Waterman with applying Hirschberg's and 
Ukkonen's algorithm and dynamic cell pruning.
Copyright (C) 2013 Matija Korpar, contributor Mile Šikić

This program is free software: you can redistribute it and/or modify
it under the terms of the GNU General Public License as published by
the Free Software Foundation, either version 3 of the License, or
(at your option) any later version.

This program is distributed in the hope that it will be useful,
but WITHOUT ANY WARRANTY; without even the implied warranty of
MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
GNU General Public License for more details.

You should have received a copy of the GNU General Public License
along with this program.  If not, see <http://www.gnu.org/licenses/>.

Contact the author by mkorpar@gmail.com.
*/

#ifdef __HIPCC__

#include <stdio.h>
#include <stdlib.h>
#include <string.h>

#include "chain.h"
#include "constants.h"
#include "cpu_module.h"
#include "cuda_utils.h"
#include "error.h"
#include "scorer.h"
#include "thread.h"
#include "threadpool.h"
#include "utils.h"

#include "score_database_gpu_short.h"

#define MAX_CPU_LEN         2000
#define CPU_WORKERS         8
#define CPU_WORKER_STEP     32

#define THREADS   128
#define BLOCKS    120

#define INT4_ZERO make_int4(0, 0, 0, 0)
#define INT4_SCORE_MIN make_int4(SCORE_MIN, SCORE_MIN, SCORE_MIN, SCORE_MIN)

typedef struct GpuDatabase {
    int card;
    int* offsets;
    int* lengths;
    int* lengthsPadded;
    hipArray* sequences;
    int* indexes;
    int* scores;
    int2* hBus;
} GpuDatabase;

typedef struct GpuDatabaseContext {
    int card;
    int length;
    int blocks;
    int* offsets;
    size_t offsetsSize;
    int* lengths;
    int* lengthsPadded;
    size_t lengthsSize;
    char4* sequences;
    int sequencesCols;
    int sequencesRows;
    size_t sequencesSize;
    int* indexes;
    size_t indexesSize;
    GpuDatabase* gpuDatabase;
} GpuDatabaseContext;

struct ShortDatabase {
    Chain** database;
    int databaseLen;
    int length;
    int* positions;
    int* order;
    int* indexes;
    int blocks;
    int sequencesRows;
    int sequencesCols;
    GpuDatabase* gpuDatabases;
    int gpuDatabasesLen;
};

typedef struct Context {
    int* scores; 
    int type;
    Chain** queries;
    int queriesLen;
    ShortDatabase* shortDatabase;
    Scorer* scorer;
    int* indexes;
    int indexesLen;
    int* cards;
    int cardsLen;
} Context;

typedef struct QueryProfile {
    int height;
    int width;
    int length;
    char4* data;
    size_t size;
} QueryProfile;

typedef struct QueryProfileGpu {
    hipArray* data;
} QueryProfileGpu;

typedef void (*ScoringFunction)(int*, int2*, int*, int*, int*, int*, int);

typedef struct KernelContext {
    int* scores;
    int type;
    ScoringFunction scoringFunction;
    QueryProfile* queryProfile;
    Chain* query;
    ShortDatabase* shortDatabase;
    Scorer* scorer;
    int* indexes;
    int indexesLen;
    int card;
} KernelContext;

typedef struct KernelContexts {
    KernelContext* contexts;
    int contextsLen;
    long long cells;
} KernelContexts;

typedef struct KernelContextCpu {
    int* scores;
    int type;
    Chain* query;
    ShortDatabase* shortDatabase;
    Scorer* scorer;
    int* indexes;
    int indexesLen;
    int* lastIndexSolvedCpu;
    int* firstIndexSolvedGpu;
    Mutex* indexSolvedMutex;
} KernelContextCpu;

typedef struct CpuWorkerContext {
    int* scores;
    int type;
    Chain* query;
    Chain** database;
    int databaseLen;
    Scorer* scorer;
    int* lastIndexSolvedCpu;
    int* firstIndexSolvedGpu;
    Mutex* indexSolvedMutex;
} CpuWorkerContext;

static __constant__ int gapOpen_;
static __constant__ int gapExtend_;

static __constant__ int rows_;
static __constant__ int rowsPadded_;
static __constant__ int width_;
static __constant__ int length_;

texture<int, 2, hipReadModeElementType> seqsTexture;
texture<char4, 2, hipReadModeElementType> qpTexture;

//******************************************************************************
// PUBLIC

extern ShortDatabase* shortDatabaseCreate(Chain** database, int databaseLen, 
    int minLen, int maxLen, int* cards, int cardsLen);

extern void shortDatabaseDelete(ShortDatabase* shortDatabase);

extern void scoreShortDatabaseGpu(int* scores, int type, Chain* query, 
    ShortDatabase* shortDatabase, Scorer* scorer, int* indexes, int indexesLen, 
    int* cards, int cardsLen, Thread* thread);

extern void scoreShortDatabasesGpu(int* scores, int type, Chain** queries, 
    int queriesLen, ShortDatabase* shortDatabase, Scorer* scorer, int* indexes, 
    int indexesLen, int* cards, int cardsLen, Thread* thread);

//******************************************************************************

//******************************************************************************
// PRIVATE

// constructor
static ShortDatabase* createDatabase(Chain** database, int databaseLen, 
    int minLen, int maxLen, int* cards, int cardsLen);

// gpu constructor thread
static void* createDatabaseGpu(void* param);

// destructor
static void deleteDatabase(ShortDatabase* database);

// scoring 
static void scoreDatabase(int* scores, int type, Chain** queries, 
    int queriesLen, ShortDatabase* shortDatabase, Scorer* scorer, int* indexes, 
    int indexesLen, int* cards, int cardsLen, Thread* thread);

static void* scoreDatabaseThread(void* param);

static void scoreDatabaseMulti(int* scores, int type,
    ScoringFunction scoringFunction, Chain** queries, int queriesLen, 
    ShortDatabase* shortDatabase, Scorer* scorer, int* indexes, int indexesLen, 
    int* cards, int cardsLen);

static void scoreDatabaseSingle(int* scores, int type,
    ScoringFunction scoringFunction, Chain** queries, int queriesLen, 
    ShortDatabase* shortDatabase, Scorer* scorer, int* indexes, int indexesLen, 
    int* cards, int cardsLen);

// cpu kernels 
static void* kernelThread(void* param);

static void* kernelsThread(void* param);

static void* kernelThreadCpu(void* param);

static void* cpuWorker(void* param);

// gpu kernels 
__global__ static void hwSolveShortGpu(int* scores, int2* hBus, int* lengths, 
    int* lengthsPadded, int* offsets, int* indexes, int block);
    
__global__ static void nwSolveShortGpu(int* scores, int2* hBus, int* lengths, 
    int* lengthsPadded, int* offsets, int* indexes, int block);

__global__ static void ovSolveShortGpu(int* scores, int2* hBus, int* lengths, 
    int* lengthsPadded, int* offsets, int* indexes, int block);

__global__ static void swSolveShortGpu(int* scores, int2* hBus, int* lengths, 
    int* lengthsPadded, int* offsets, int* indexes, int block);
    
// query profile
static QueryProfile* createQueryProfile(Chain* query, Scorer* scorer);

static void deleteQueryProfile(QueryProfile* queryProfile);

static QueryProfileGpu* createQueryProfileGpu(QueryProfile* queryProfile);

static void deleteQueryProfileGpu(QueryProfileGpu* queryProfileGpu);

// utils
static int int2CmpY(const void* a_, const void* b_);

//******************************************************************************

//******************************************************************************
// PUBLIC

//------------------------------------------------------------------------------
// CONSTRUCTOR, DESTRUCTOR

extern ShortDatabase* shortDatabaseCreate(Chain** database, int databaseLen, 
    int minLen, int maxLen, int* cards, int cardsLen) {
    return createDatabase(database, databaseLen, minLen, maxLen, cards, cardsLen);
}
    
extern void shortDatabaseDelete(ShortDatabase* shortDatabase) {
    deleteDatabase(shortDatabase);
}

extern size_t shortDatabaseGpuMemoryConsumption(Chain** database,
    int databaseLen, int minLen, int maxLen) {

    int length = 0;
    int maxHeight = 0;

    for (int i = 0; i < databaseLen; ++i) {

        const int n = chainGetLength(database[i]);
        
        if (n >= minLen && n < maxLen) {
            length++;
            maxHeight = max(maxHeight, n);
        }
    }

    if (length == 0) {
        return 0;
    }

    maxHeight = (maxHeight >> 2) + ((maxHeight & 3) > 0);

    int sequencesCols = THREADS * BLOCKS;

    int blocks = length / sequencesCols + (length % sequencesCols > 0);
    int hBusHeight = maxHeight * 4;

    //##########################################################################

    const int bucketDiff = 32;
    int bucketsLen = maxLen / bucketDiff + (maxLen % bucketDiff > 0);

    int* buckets = (int*) malloc(bucketsLen * sizeof(int));
    memset(buckets, 0, bucketsLen * sizeof(int));

    for (int i = 0; i < databaseLen; ++i) {

        const int n = chainGetLength(database[i]);
        
        if (n >= minLen && n < maxLen) {
            buckets[n >> 5]++;
        }
    }

    int sequencesRows = 0;
    for (int i = 0, j = 0; i < bucketsLen; ++i) {
        
        j += buckets[i];

        int d = j / sequencesCols;
        int r = j % sequencesCols;

        sequencesRows += d * ((i + 1) * (bucketDiff / 4));
        j = r;

        if (i == bucketsLen - 1 && j > 0) {
            sequencesRows += ((i + 1) * (bucketDiff / 4));
        }
    }

    free(buckets);

    /*
    int* lengths = (int*) malloc(length * sizeof(int));

    for (int i = 0, j = 0; i < databaseLen; ++i) {

        const int n = chainGetLength(database[i]);
        
        if (n >= minLen && n < maxLen) {
            lengths[j++] = n;
        }
    }

    qsort(lengths, length, sizeof(int), intCmp);
    

    int sequencesRows = 0;

    for (int i = sequencesCols - 1; i < length; i += sequencesCols) {
        int n = lengths[i];
        sequencesRows += (n >> 2) + ((n & 3) > 0);
    }

    if (length % sequencesCols != 0) {
        sequencesRows += maxHeight;
    }

  
  free(lengths);
    */

    //##########################################################################

    size_t hBusSize = sequencesCols * hBusHeight * sizeof(int2);
    size_t offsetsSize = blocks * sizeof(int);
    size_t lengthsSize = blocks * sequencesCols * sizeof(int);
    size_t sequencesSize = sequencesRows * sequencesCols * sizeof(char4);
    size_t scoresSize = length * sizeof(int);
    size_t indexesSize = length * sizeof(int);

    size_t memory = offsetsSize + 2 * lengthsSize + sequencesSize + 
        indexesSize + scoresSize + hBusSize;

    return memory;
}

//------------------------------------------------------------------------------

//------------------------------------------------------------------------------
// DATABASE SCORING

extern void scoreShortDatabaseGpu(int* scores, int type, Chain* query, 
    ShortDatabase* shortDatabase, Scorer* scorer, int* indexes, int indexesLen, 
    int* cards, int cardsLen, Thread* thread) {
    scoreDatabase(scores, type, &query, 1, shortDatabase, scorer, indexes, 
        indexesLen, cards, cardsLen, thread);
}

extern void scoreShortDatabasesGpu(int* scores, int type, Chain** queries, 
    int queriesLen, ShortDatabase* shortDatabase, Scorer* scorer, int* indexes, 
    int indexesLen, int* cards, int cardsLen, Thread* thread) {
    scoreDatabase(scores, type, queries, queriesLen, shortDatabase, scorer,
        indexes, indexesLen, cards, cardsLen, thread);
}

//------------------------------------------------------------------------------

//******************************************************************************

//******************************************************************************
// PRIVATE

//------------------------------------------------------------------------------
// CONSTRUCTOR, DESTRUCTOR 

static ShortDatabase* createDatabase(Chain** database, int databaseLen, 
    int minLen, int maxLen, int* cards, int cardsLen) {
    
    ASSERT(cardsLen > 0, "no GPUs available");

    //**************************************************************************
    // FILTER DATABASE AND REMEBER ORDER
    
    int length = 0;
    
    for (int i = 0; i < databaseLen; ++i) {
    
        const int n = chainGetLength(database[i]);
        
        if (n >= minLen && n < maxLen) {
            length++;
        }
    }
    
    if (length == 0) {
        return NULL;
    }
    
    int2* orderPacked = (int2*) malloc(length * sizeof(int2));

    for (int i = 0, j = 0; i < databaseLen; ++i) {
    
        const int n = chainGetLength(database[i]);
        
        if (n >= minLen && n < maxLen) {
            orderPacked[j].x = i;
            orderPacked[j].y = n;
            j++;
        }
    }
    
    qsort(orderPacked, length, sizeof(int2), int2CmpY);
    
    LOG("Short database length: %d", length);

    //**************************************************************************

    //**************************************************************************
    // CALCULATE GRID DIMENSIONS
    
    int sequencesCols = THREADS * BLOCKS;
    int sequencesRows = 0;

    int blocks = 0;
    for (int i = sequencesCols - 1; i < length; i += sequencesCols) {
        int n = chainGetLength(database[orderPacked[i].x]);
        sequencesRows += (n >> 2) + ((n & 3) > 0);
        blocks++;
    }
    
    if (length % sequencesCols != 0) {
        int n = chainGetLength(database[orderPacked[length - 1].x]);
        sequencesRows += (n >> 2) + ((n & 3) > 0);
        blocks++;
    }
    
    LOG("Short database grid: %d(%d)x%d", sequencesRows, blocks, sequencesCols);
    
    //**************************************************************************
    
    //**************************************************************************
    // INIT STRUCTURES
    
    size_t offsetsSize = blocks * sizeof(int);
    int* offsets = (int*) malloc(offsetsSize);
    
    size_t lengthsSize = blocks * sequencesCols * sizeof(int);
    int* lengths = (int*) malloc(lengthsSize);
    int* lengthsPadded = (int*) malloc(lengthsSize);
    
    size_t sequencesSize = sequencesRows * sequencesCols * sizeof(char4);
    char4* sequences = (char4*) malloc(sequencesSize);
    
    //***********f***************************************************************

    //**************************************************************************
    // CREATE GRID
    
    // tmp
    size_t sequenceSize = chainGetLength(database[orderPacked[length - 1].x]) + 4;
    char* sequence = (char*) malloc(sequenceSize);

    offsets[0] = 0;
    for(int i = 0, j = 0, cx = 0, cy = 0; i < length; i++){

        //get the sequence and its length
        Chain* chain = database[orderPacked[i].x];
        int n = chainGetLength(chain);    
        
        lengths[j * sequencesCols + cx] = n;
        
        chainCopyCodes(chain, sequence);
        memset(sequence + n, 127, 4 * sizeof(char));

        int n4 = (n >> 2) + ((n & 3) > 0);

        lengthsPadded[j * sequencesCols + cx] = n4;
        
        char4* ptr = sequences + cy * sequencesCols + cx;
        for(int k = 0; k < n; k += 4){
            ptr->x = sequence[k];
            ptr->y = sequence[k + 1];
            ptr->z = sequence[k + 2];
            ptr->w = sequence[k + 3];
            ptr += sequencesCols;
        }

        cx++;
        
        if(cx == sequencesCols){
            offsets[j + 1] = offsets[j] + n4;
            cx = 0;
            cy += n4;
            j++;
        }
    }
    
    free(sequence);
    
    //**************************************************************************
    
    //**************************************************************************
    // CREATE POSITION ARRAY
    
    int* positions = (int*) malloc(databaseLen * sizeof(int));

    for (int i = 0; i < databaseLen; ++i) {
        positions[i] = -1;
    }
    
    for (int i = 0; i < length; ++i) {
        positions[orderPacked[i].x] = i;
    }
    
    //**************************************************************************
    
    //**************************************************************************
    // CREATE ORDER ARRAY
    
    size_t orderSize = length * sizeof(int);
    int* order = (int*) malloc(orderSize);

    for (int i = 0; i < length; ++i) {
        order[i] = orderPacked[i].x;
    }
     
    //**************************************************************************
    
    //**************************************************************************
    // CREATE DEFAULT INDEXES
    
    size_t indexesSize = length * sizeof(int);
    int* indexes = (int*) malloc(indexesSize);

    for (int i = 0; i < length; ++i) {
        indexes[i] = i;
    }
     
    //**************************************************************************

    //**************************************************************************
    // CREATE GPU DATABASES
    
    size_t gpuDatabasesSize = cardsLen * sizeof(GpuDatabase);
    GpuDatabase* gpuDatabases = (GpuDatabase*) malloc(gpuDatabasesSize);

    GpuDatabaseContext* contexts = 
        (GpuDatabaseContext*) malloc(cardsLen * sizeof(GpuDatabaseContext));

    Thread* threads = (Thread*) malloc(cardsLen * sizeof(Thread));

    for (int i = 0; i < cardsLen; ++i) {

        GpuDatabaseContext* context = &(contexts[i]);

        context->card = cards[i];
        context->length = length;
        context->blocks = blocks;
        context->offsets = offsets;
        context->offsetsSize = offsetsSize;
        context->lengths = lengths;
        context->lengthsPadded = lengthsPadded;
        context->lengthsSize = lengthsSize;
        context->sequences = sequences;
        context->sequencesCols = sequencesCols;
        context->sequencesRows = sequencesRows;
        context->sequencesSize = sequencesSize;
        context->indexes = indexes;
        context->indexesSize = indexesSize;
        context->gpuDatabase = gpuDatabases + i;
    }

    for (int i = 1; i < cardsLen; ++i) {
        threadCreate(&(threads[i]), createDatabaseGpu, (void*) &(contexts[i]));
    }

    createDatabaseGpu((void*) &(contexts[0]));

    for (int i = 1; i < cardsLen; ++i) {
        threadJoin(threads[i]);
    }

    free(contexts);
    free(threads);

    //**************************************************************************
    
    //**************************************************************************
    // CLEAN MEMORY

    free(orderPacked);
    free(offsets);
    free(lengths);
    free(lengthsPadded);
    free(sequences);

    //**************************************************************************
    
    size_t shortDatabaseSize = sizeof(struct ShortDatabase);
    ShortDatabase* shortDatabase = (ShortDatabase*) malloc(shortDatabaseSize);
    
    shortDatabase->database = database;
    shortDatabase->databaseLen = databaseLen;
    shortDatabase->length = length;
    shortDatabase->positions = positions;
    shortDatabase->order = order;
    shortDatabase->indexes = indexes;
    shortDatabase->blocks = blocks;
    shortDatabase->sequencesRows = sequencesRows;
    shortDatabase->sequencesCols = sequencesCols;
    shortDatabase->gpuDatabases = gpuDatabases;
    shortDatabase->gpuDatabasesLen = cardsLen;
    
    return shortDatabase;
}

static void* createDatabaseGpu(void* param) {

    GpuDatabaseContext* context = (GpuDatabaseContext*) param;

    int card = context->card;
    int length = context->length;
    int blocks = context->blocks;
    int* offsets = context->offsets;
    size_t offsetsSize = context->offsetsSize;
    int* lengths = context->lengths;
    int* lengthsPadded = context->lengthsPadded;
    size_t lengthsSize = context->lengthsSize;
    char4* sequences = context->sequences;
    int sequencesCols = context->sequencesCols;
    int sequencesRows = context->sequencesRows;
    size_t sequencesSize = context->sequencesSize;
    int* indexes = context->indexes;
    size_t indexesSize = context->indexesSize;
    GpuDatabase* gpuDatabase = context->gpuDatabase;

    CUDA_SAFE_CALL(hipSetDevice(card));

    int* offsetsGpu;
    CUDA_SAFE_CALL(hipMalloc(&offsetsGpu, offsetsSize));
    CUDA_SAFE_CALL(hipMemcpy(offsetsGpu, offsets, offsetsSize, TO_GPU));
    
    int* lengthsGpu;
    CUDA_SAFE_CALL(hipMalloc(&lengthsGpu, lengthsSize));
    CUDA_SAFE_CALL(hipMemcpy(lengthsGpu, lengths, lengthsSize, TO_GPU));

    int* lengthsPaddedGpu;
    CUDA_SAFE_CALL(hipMalloc(&lengthsPaddedGpu, lengthsSize));
    CUDA_SAFE_CALL(hipMemcpy(lengthsPaddedGpu, lengthsPadded, lengthsSize, TO_GPU));
    
    hipArray* sequencesGpu;
    hipChannelFormatDesc channel = seqsTexture.channelDesc;
    CUDA_SAFE_CALL(hipMallocArray(&sequencesGpu, &channel, sequencesCols, sequencesRows)); 
    CUDA_SAFE_CALL(hipMemcpyToArray(sequencesGpu, 0, 0, sequences, sequencesSize, TO_GPU));
    CUDA_SAFE_CALL(hipBindTextureToArray(seqsTexture, sequencesGpu));

    int* indexesGpu;
    CUDA_SAFE_CALL(hipMalloc(&indexesGpu, indexesSize));
    CUDA_SAFE_CALL(hipMemcpy(indexesGpu, indexes, indexesSize, TO_GPU));
    
    // additional structures

    size_t scoresSize = length * sizeof(int);
    int* scoresGpu;
    CUDA_SAFE_CALL(hipMalloc(&scoresGpu, scoresSize));

    int2* hBusGpu;
    int hBusHeight = (sequencesRows - offsets[blocks - 1]) * 4;
    size_t hBusSize = sequencesCols * hBusHeight * sizeof(int2);
    CUDA_SAFE_CALL(hipMalloc(&hBusGpu, hBusSize));

    gpuDatabase->card = card;
    gpuDatabase->offsets = offsetsGpu;
    gpuDatabase->lengths = lengthsGpu;
    gpuDatabase->lengthsPadded = lengthsPaddedGpu;
    gpuDatabase->sequences = sequencesGpu;
    gpuDatabase->indexes = indexesGpu;
    gpuDatabase->scores = scoresGpu;
    gpuDatabase->hBus = hBusGpu;
    
#ifdef DEBUG
    size_t memory = offsetsSize + 2 * lengthsSize + sequencesSize + 
        indexesSize + scoresSize + hBusSize;

    LOG("Short database using %.2lfMBs on card %d", memory / 1024.0 / 1024.0, card);
#endif

    return NULL;
}

static void deleteDatabase(ShortDatabase* database) {

    if (database == NULL) {
        return;
    }
    
    for (int i = 0; i < database->gpuDatabasesLen; ++i) {
    
        GpuDatabase* gpuDatabase = &(database->gpuDatabases[i]);
        
        CUDA_SAFE_CALL(hipSetDevice(gpuDatabase->card));

        CUDA_SAFE_CALL(hipFree(gpuDatabase->offsets));
        CUDA_SAFE_CALL(hipFree(gpuDatabase->lengths));
        CUDA_SAFE_CALL(hipFree(gpuDatabase->lengthsPadded));
        CUDA_SAFE_CALL(hipFreeArray(gpuDatabase->sequences));
        CUDA_SAFE_CALL(hipFree(gpuDatabase->indexes));
        CUDA_SAFE_CALL(hipFree(gpuDatabase->scores));
        CUDA_SAFE_CALL(hipFree(gpuDatabase->hBus));

        CUDA_SAFE_CALL(hipUnbindTexture(seqsTexture));
    }

    free(database->gpuDatabases);
    free(database->positions);
    free(database->order);
    free(database->indexes);

    free(database);
}

//------------------------------------------------------------------------------

//------------------------------------------------------------------------------
// ENTRY 

static void scoreDatabase(int* scores, int type, Chain** queries, 
    int queriesLen, ShortDatabase* shortDatabase, Scorer* scorer, int* indexes, 
    int indexesLen, int* cards, int cardsLen, Thread* thread) {
    
    ASSERT(cardsLen > 0, "no GPUs available");
    
    Context* param = (Context*) malloc(sizeof(Context));
    
    param->scores = scores;
    param->type = type;
    param->queries = queries;
    param->queriesLen = queriesLen;
    param->shortDatabase = shortDatabase;
    param->scorer = scorer;
    param->indexes = indexes;
    param->indexesLen = indexesLen;
    param->cards = cards;
    param->cardsLen = cardsLen;
    
    if (thread == NULL) {
        scoreDatabaseThread(param);
    } else {
        threadCreate(thread, scoreDatabaseThread, (void*) param);
    }
}

//------------------------------------------------------------------------------

//------------------------------------------------------------------------------
// DATABASE SCORING

static void* scoreDatabaseThread(void* param) {

    Context* context = (Context*) param;
    
    int* scores = context->scores;
    int type = context->type;
    Chain** queries = context->queries;
    int queriesLen = context->queriesLen;
    ShortDatabase* shortDatabase = context->shortDatabase;
    Scorer* scorer = context->scorer;
    int* indexes = context->indexes;
    int indexesLen = context->indexesLen;
    int* cards = context->cards;
    int cardsLen = context->cardsLen;

    if (shortDatabase == NULL) {
        return NULL;
    }

    //**************************************************************************
    // CREATE NEW INDEXES ARRAY IF NEEDED
    
    int* newIndexes = NULL;
    int newIndexesLen = 0;

    int deleteIndexes;

    if (indexes != NULL) {

        // translate and filter indexes, also make sure that indexes are 
        // sorted by size 
    
        int length = shortDatabase->length;
        int databaseLen = shortDatabase->databaseLen;
        int* positions = shortDatabase->positions;
        
        char* solveMask = (char*) malloc(length * sizeof(char));
        memset(solveMask, 0, length);
        
        newIndexesLen = 0;
        for (int i = 0; i < indexesLen; ++i) {
            
            int idx = indexes[i];
            if (idx < 0 || idx > databaseLen || positions[idx] == -1) {
                continue;
            }
            
            solveMask[positions[idx]] = 1;
            newIndexesLen++;
        }
        
        newIndexes = (int*) malloc(newIndexesLen * sizeof(int));
        
        for (int i = 0, j = 0; i < length; ++i) {
            if (solveMask[i]) {
                newIndexes[j++] = i;
            }
        }
        
        free(solveMask);

        deleteIndexes = 1;

    } else {
        // load prebuilt defaults
        newIndexes = shortDatabase->indexes;
        newIndexesLen = shortDatabase->length;
        deleteIndexes = 0;
    }
    
    //**************************************************************************

    //**************************************************************************
    // CHOOSE SOLVING FUNCTION
    
    ScoringFunction function;
    switch (type) {
    case SW_ALIGN: 
        function = swSolveShortGpu;
        break;
    case NW_ALIGN: 
        function = nwSolveShortGpu;
        break;
    case HW_ALIGN:
        function = hwSolveShortGpu;
        break;
    case OV_ALIGN:
        function = ovSolveShortGpu;
        break;
    default:
        ERROR("Wrong align type");
    }
    
    //**************************************************************************

    //**************************************************************************
    // SCORE MULTITHREADED

    if (queriesLen < cardsLen) {
        scoreDatabaseMulti(scores, type, function, queries, queriesLen, 
            shortDatabase, scorer, newIndexes, newIndexesLen, cards, cardsLen);
    } else {
        scoreDatabaseSingle(scores, type, function, queries, queriesLen, 
            shortDatabase, scorer, newIndexes, newIndexesLen, cards, cardsLen);
    }
    
    //**************************************************************************

    //**************************************************************************
    // CLEAN MEMORY

    if (deleteIndexes) {
        free(newIndexes);
    }

    free(param);
    
    //**************************************************************************
    
    return NULL;
}

static void scoreDatabaseMulti(int* scores, int type, 
    ScoringFunction scoringFunction, Chain** queries, int queriesLen, 
    ShortDatabase* shortDatabase, Scorer* scorer, int* indexes, int indexesLen, 
    int* cards, int cardsLen) {
    
    //**************************************************************************
    // CREATE QUERY PROFILES
    
    size_t profilesSize = queriesLen * sizeof(QueryProfile*);
    QueryProfile** profiles = (QueryProfile**) malloc(profilesSize);
    
    for (int i = 0; i < queriesLen; ++i) {
        profiles[i] = createQueryProfile(queries[i], scorer);
    }
    
    //**************************************************************************
    
    //**************************************************************************
    // CREATE BALANCING DATA

    Chain** database = shortDatabase->database;
    int* order = shortDatabase->order;
    int sequencesCols = shortDatabase->sequencesCols;
    
    int blocks = indexesLen / sequencesCols + ((indexesLen % sequencesCols) > 0);

    size_t weightsSize = blocks * sizeof(int);
    int* weights = (int*) malloc(weightsSize);
    memset(weights, 0, weightsSize);

    for (int i = 0, j = 0; i < indexesLen; ++i) {

        weights[j] += chainGetLength(database[order[indexes[i]]]);

        if ((i + 1) % sequencesCols == 0) {
            j++;
        }
    }

    //**************************************************************************

    //**************************************************************************
    // SCORE MULTICARDED
    
    int contextsLen = cardsLen * queriesLen;
    size_t contextsSize = contextsLen * sizeof(KernelContext);
    KernelContext* contexts = (KernelContext*) malloc(contextsSize);
    
    size_t tasksSize = contextsLen * sizeof(Thread);
    Thread* tasks = (Thread*) malloc(tasksSize);

    int databaseLen = shortDatabase->databaseLen;
    
    int cardsChunk = cardsLen / queriesLen;
    int cardsAdd = cardsLen % queriesLen;
    int cardsOff = 0;

    int* idxChunksOff = (int*) malloc(cardsLen * sizeof(int));
    int* idxChunksLens = (int*) malloc(cardsLen * sizeof(int));
    int idxChunksLen = 0;
    int idxLastFix = (sequencesCols - indexesLen % sequencesCols) % sequencesCols;

    int length = 0;

    for (int i = 0, k = 0; i < queriesLen; ++i) {

        int cCardsLen = cardsChunk + (i < cardsAdd);
        int* cCards = cards + cardsOff;
        cardsOff += cCardsLen;
        
        QueryProfile* queryProfile = profiles[i];

        int chunks = min(cCardsLen, blocks);
        if (chunks != idxChunksLen) {
            weightChunkArray(idxChunksOff, idxChunksLens, &idxChunksLen, 
                weights, blocks, chunks);
        }
        
        for (int j = 0; j < idxChunksLen; ++j, ++k) {
        
            int off = idxChunksOff[j] * sequencesCols;
            int len = idxChunksLens[j] * sequencesCols;
            if (j == idxChunksLen - 1) {
                len -= idxLastFix;
            }
            
            contexts[k].scores = scores + i * databaseLen;
            contexts[k].type = type;
            contexts[k].scoringFunction = scoringFunction;
            contexts[k].queryProfile = queryProfile;
            contexts[k].query = queries[i];
            contexts[k].shortDatabase = shortDatabase;
            contexts[k].scorer = scorer;
            contexts[k].indexes = indexes + off;
            contexts[k].indexesLen = len;
            contexts[k].card = cCards[j];

            threadCreate(&(tasks[k]), kernelThread, &(contexts[k]));
            length++;
        }
    }
    
    for (int i = 0; i < length; ++i) {
        threadJoin(tasks[i]);
    }

    free(tasks);
    free(contexts);

    //**************************************************************************
    
    //**************************************************************************
    // CLEAN MEMORY

    for (int i = 0; i < queriesLen; ++i) {
        deleteQueryProfile(profiles[i]);
    }

    free(profiles);
    free(weights);
    free(idxChunksOff);
    free(idxChunksLens);
    
    //**************************************************************************
}

static void scoreDatabaseSingle(int* scores, int type, 
    ScoringFunction scoringFunction, Chain** queries, int queriesLen, 
    ShortDatabase* shortDatabase, Scorer* scorer, int* indexes, int indexesLen, 
    int* cards, int cardsLen) {

    //**************************************************************************
    // CREATE CONTEXTS
    
    size_t contextsSize = cardsLen * sizeof(KernelContext);
    KernelContexts* contexts = (KernelContexts*) malloc(contextsSize);
    
    for (int i = 0; i < cardsLen; ++i) {
        size_t size = queriesLen * sizeof(KernelContext);
        contexts[i].contexts = (KernelContext*) malloc(size);
        contexts[i].contextsLen = 0;
        contexts[i].cells = 0;
    }
    
    //**************************************************************************    
    
    //**************************************************************************
    // SCORE MULTITHREADED
    
    size_t tasksSize = cardsLen * sizeof(Thread);
    Thread* tasks = (Thread*) malloc(tasksSize);
    
    int databaseLen = shortDatabase->databaseLen;
    
    // balance tasks by round roobin, cardsLen is pretty small (CUDA cards)
    for (int i = 0; i < queriesLen; ++i) {
        
        int minIdx = 0;
        long long minVal = contexts[0].cells;
        for (int j = 1; j < cardsLen; ++j) {
            if (contexts[j].cells < minVal) {
                minVal = contexts[j].cells;
                minIdx = j;
            }
        }
        
        KernelContext context;
        context.scores = scores + i * databaseLen;
        context.type = type;
        context.scoringFunction = scoringFunction;
        context.queryProfile = NULL;
        context.query = queries[i];
        context.shortDatabase = shortDatabase;
        context.scorer = scorer;
        context.indexes = indexes;
        context.indexesLen = indexesLen;
        context.card = cards[minIdx];

        contexts[minIdx].contexts[contexts[minIdx].contextsLen++] = context;
        contexts[minIdx].cells += chainGetLength(queries[i]);
    }
    
    for (int i = 0; i < cardsLen; ++i) {
        threadCreate(&(tasks[i]), kernelsThread, &(contexts[i]));
    }

    for (int i = 0; i < cardsLen; ++i) {
        threadJoin(tasks[i]);
    }
    free(tasks);

    //**************************************************************************
    
    //**************************************************************************
    // CLEAN MEMORY

    for (int i = 0; i < cardsLen; ++i) {
        free(contexts[i].contexts);
    }
    free(contexts);

    //**************************************************************************
}

//------------------------------------------------------------------------------

//------------------------------------------------------------------------------
// CPU KERNELS

static void* kernelsThread(void* param) {

    KernelContexts* context = (KernelContexts*) param;

    KernelContext* contexts = context->contexts;
    int contextsLen = context->contextsLen;

    for (int i = 0; i < contextsLen; ++i) {
    
        Chain* query = contexts[i].query;
        Scorer* scorer = contexts[i].scorer;
        int card = contexts[i].card;
        
        int currentCard;
        CUDA_SAFE_CALL(hipGetDevice(&currentCard));
        if (currentCard != card) {
            CUDA_SAFE_CALL(hipSetDevice(card));
        }
    
        contexts[i].queryProfile = createQueryProfile(query, scorer);
        
        kernelThread(&(contexts[i]));
        
        deleteQueryProfile(contexts[i].queryProfile);
    }
    
    return NULL;
}

static void* kernelThread(void* param) {

    KernelContext* context = (KernelContext*) param;
    
    int* scores = context->scores;
    int type = context->type;
    ScoringFunction scoringFunction = context->scoringFunction;
    Chain* query = context->query;
    QueryProfile* queryProfile = context->queryProfile;
    ShortDatabase* shortDatabase = context->shortDatabase;
    Scorer* scorer = context->scorer;
    int* indexes = context->indexes;
    int indexesLen = context->indexesLen;
    int card = context->card;
    
    //**************************************************************************
    // FIND DATABASE
    
    GpuDatabase* gpuDatabases = shortDatabase->gpuDatabases;
    int gpuDatabasesLen = shortDatabase->gpuDatabasesLen;
    
    GpuDatabase* gpuDatabase = NULL;
    
    for (int i = 0; i < gpuDatabasesLen; ++i) {
        if (gpuDatabases[i].card == card) {
            gpuDatabase = &(gpuDatabases[i]);
            break;
        }
    }

    ASSERT(gpuDatabase != NULL, "Short database not available on card %d", card);

    //**************************************************************************
    
    //**************************************************************************
    // CUDA SETUP
    
    int currentCard;
    CUDA_SAFE_CALL(hipGetDevice(&currentCard));
    if (currentCard != card) {
        CUDA_SAFE_CALL(hipSetDevice(card));
    }
    
    //**************************************************************************
    
    //**************************************************************************
    // FIX INDEXES
    
    int deleteIndexes;
    int* indexesGpu;
    
    if (indexesLen == shortDatabase->length) {
        indexes = shortDatabase->indexes;
        indexesLen = shortDatabase->length;
        indexesGpu = gpuDatabase->indexes;
        deleteIndexes = 0;
    } else {
        size_t indexesSize = indexesLen * sizeof(int);
        CUDA_SAFE_CALL(hipMalloc(&indexesGpu, indexesSize));
        CUDA_SAFE_CALL(hipMemcpy(indexesGpu, indexes, indexesSize, TO_GPU));
        deleteIndexes = 1;
    }

    //**************************************************************************
    
    //**************************************************************************
    // PREPARE GPU
    
    QueryProfileGpu* queryProfileGpu = createQueryProfileGpu(queryProfile);
    
    int gapOpen = scorerGetGapOpen(scorer);
    int gapExtend = scorerGetGapExtend(scorer);
    int rows = queryProfile->length;
    int rowsGpu = queryProfile->height * 4;
    int sequencesCols = shortDatabase->sequencesCols;
    
    CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(gapOpen_), &gapOpen, sizeof(int)));
    CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(gapExtend_), &gapExtend, sizeof(int)));
    CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(rows_), &rows, sizeof(int)));
    CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(rowsPadded_), &rowsGpu, sizeof(int)));
    CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(width_), &sequencesCols, sizeof(int)));
    CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(length_), &indexesLen, sizeof(int)));
    
    //**************************************************************************

    //**************************************************************************
    // PREPARE CPU

    int lastIndexSolvedCpu = 0;
    int firstIndexSolvedGpu = INT_MAX;

    Mutex indexSolvedMutex;
    mutexCreate(&indexSolvedMutex);
    
    KernelContextCpu* contextCpu = (KernelContextCpu*) malloc(sizeof(KernelContextCpu));

    contextCpu->scores = scores;
    contextCpu->type = type;
    contextCpu->query = query;
    contextCpu->shortDatabase = shortDatabase;
    contextCpu->scorer = scorer;
    contextCpu->indexes = indexes;
    contextCpu->indexesLen = indexesLen;
    contextCpu->lastIndexSolvedCpu = &lastIndexSolvedCpu;
    contextCpu->firstIndexSolvedGpu = &firstIndexSolvedGpu;
    contextCpu->indexSolvedMutex = &indexSolvedMutex;
    
    //**************************************************************************

    //**************************************************************************
    // SOLVE

    Thread thread;
    threadCreate(&thread, kernelThreadCpu, contextCpu);

    int blocks = shortDatabase->blocks;
    
    int* offsetsGpu = gpuDatabase->offsets;
    int* lengthsGpu = gpuDatabase->lengths;
    int* lengthsPaddedGpu = gpuDatabase->lengthsPadded;
    int* scoresGpu = gpuDatabase->scores;
    int2* hBusGpu = gpuDatabase->hBus;
    
    TIMER_START("Short GPU solving: %d", indexesLen);

    for (int i = blocks - 1; i >= 0; --i) {

        if (sequencesCols * i > indexesLen) {
            continue;
        }

        // wait for iteration to finish
        CUDA_SAFE_CALL(hipDeviceSynchronize());

        int firstIdx = sequencesCols * i;
        int lastIdx = min(sequencesCols * (i + 1) - 1, indexesLen - 1);

        // multithreaded, chech mutexes
        mutexLock(&indexSolvedMutex);

        // indexes already solved
        if (lastIdx < lastIndexSolvedCpu) {
            mutexUnlock(&indexSolvedMutex);
            break;
        }

        firstIndexSolvedGpu = min(firstIdx, firstIndexSolvedGpu);

        mutexUnlock(&indexSolvedMutex);

        scoringFunction<<<BLOCKS, THREADS>>>(scoresGpu, hBusGpu, lengthsGpu, 
            lengthsPaddedGpu, offsetsGpu, indexesGpu, i);
    }

    TIMER_STOP;

    threadJoin(thread);

    //**************************************************************************
    
    //**************************************************************************
    // SAVE RESULTS

    int length = shortDatabase->length;
    
    size_t scoresSize = length * sizeof(int);
    int* scoresCpu = (int*) malloc(scoresSize);

    CUDA_SAFE_CALL(hipMemcpy(scoresCpu, scoresGpu, scoresSize, FROM_GPU));

    int* order = shortDatabase->order;
    
    for (int i = firstIndexSolvedGpu; i < indexesLen; ++i) {
        scores[order[indexes[i]]] = scoresCpu[indexes[i]];
    }
    
    free(scoresCpu);
                
    //**************************************************************************

    //**************************************************************************
    // CLEAN MEMORY
    
    deleteQueryProfileGpu(queryProfileGpu);
    
    if (deleteIndexes) {
        CUDA_SAFE_CALL(hipFree(indexesGpu));
    }

    mutexDelete(&indexSolvedMutex);
    free(contextCpu);

    //**************************************************************************
    
    return NULL;
}

static void* kernelThreadCpu(void* param) {

    int i;

    KernelContextCpu* context = (KernelContextCpu*) param;

    int* scores = context->scores;
    int type = context->type;
    Chain* query = context->query;
    ShortDatabase* shortDatabase = context->shortDatabase;
    Scorer* scorer = context->scorer;
    int* indexes = context->indexes;
    int indexesLen = context->indexesLen;
    int* lastIndexSolvedCpu = context->lastIndexSolvedCpu;
    int* firstIndexSolvedGpu = context->firstIndexSolvedGpu;
    Mutex* indexSolvedMutex = context->indexSolvedMutex;

    int* order = shortDatabase->order;

    TIMER_START("Short CPU solving");

    //**************************************************************************
    // CREATE DATABASE
    

    Chain** database = (Chain**) malloc(indexesLen * sizeof(Chain*));
    int databaseLen = 0;

    for (i = 0; i < indexesLen; ++i) {

        Chain* chain = shortDatabase->database[order[indexes[i]]];

        if (chainGetLength(chain) > MAX_CPU_LEN) {
            break;
        }

        database[i] = chain;
        databaseLen++;
    }

    LOG("Max CPU chains: %d", databaseLen);

    //**************************************************************************

    //**************************************************************************
    // SOLVE

    int* scoresCpu = (int*) malloc(databaseLen * sizeof(int));

    int workers = min(CPU_WORKERS, databaseLen);

    CpuWorkerContext* contexts = (CpuWorkerContext*) malloc(workers * sizeof(CpuWorkerContext));
    Thread* tasks = (Thread*) malloc(workers * sizeof(Thread));

    for (i = 0; i < workers; ++i) {

        contexts[i].scores = scoresCpu;
        contexts[i].type = type;
        contexts[i].query = query;
        contexts[i].database = database;
        contexts[i].databaseLen = databaseLen;
        contexts[i].scorer = scorer;
        contexts[i].lastIndexSolvedCpu = lastIndexSolvedCpu;
        contexts[i].firstIndexSolvedGpu = firstIndexSolvedGpu;
        contexts[i].indexSolvedMutex = indexSolvedMutex;

        threadCreate(&(tasks[i]), cpuWorker, &(contexts[i]));
    }
    
    for (i = 0; i < workers; ++i) {
        threadJoin(tasks[i]);
    }

    free(tasks);
    free(contexts);

    //**************************************************************************

    //**************************************************************************
    // SAVE RESULTS

    LOG("CPU solved %d chains", *lastIndexSolvedCpu);

    for (int i = 0; i <= *lastIndexSolvedCpu; ++i) {
        scores[order[indexes[i]]] = scoresCpu[i];
    }
    
    //**************************************************************************

    //**************************************************************************
    // CLEAN MEMORY

    free(scoresCpu);
    free(database);

    //**************************************************************************

    TIMER_STOP;

    return NULL;
}

static void* cpuWorker(void* param) {

    CpuWorkerContext* context = (CpuWorkerContext*) param;

    int* scores = context->scores;
    int type = context->type;
    Chain* query = context->query;
    Chain** database = context->database;
    int databaseLen = context->databaseLen;
    Scorer* scorer = context->scorer;
    int* lastIndexSolvedCpu = context->lastIndexSolvedCpu;
    int* firstIndexSolvedGpu = context->firstIndexSolvedGpu;
    Mutex* indexSolvedMutex = context->indexSolvedMutex;

    while (1) {

        mutexLock(indexSolvedMutex);

        int start = max(0, *lastIndexSolvedCpu);
        int length = min(CPU_WORKER_STEP, databaseLen - start);

        if (start >= databaseLen || start > *firstIndexSolvedGpu - THREADS * BLOCKS) {
            mutexUnlock(indexSolvedMutex);
            break;
        }

        *lastIndexSolvedCpu += length;

        mutexUnlock(indexSolvedMutex);

        scoreDatabaseCpu(scores + start, type, query, database + start, length, scorer);
    }

    return NULL;
}

//------------------------------------------------------------------------------

//------------------------------------------------------------------------------
// GPU KERNELS

__device__ static int gap(int index) {
    return (-gapOpen_ - index * gapExtend_) * (index >= 0);
}

__global__ static void hwSolveShortGpu(int* scores, int2* hBus, int* lengths, 
    int* lengthsPadded, int* offsets, int* indexes, int block) {

    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    
    if (tid + block * width_ >= length_) {
        return;
    }
    
    int id = indexes[tid + block * width_];
    int cols = lengthsPadded[id];
    int realCols = lengths[id];
    
    int colOff = id % width_;
    int rowOff = offsets[id / width_];
    
    int score = SCORE_MIN;
    
    int4 scrUp;
    int4 affUp;
    int4 mchUp;
    
    int4 scrDown;
    int4 affDown;
    int4 mchDown;
    
    int2 wBus;
    int del;
    
    int lastRow = rows_ - 1;
    
    for (int j = 0; j < cols * 4; ++j) {
        hBus[j * width_ + tid] = make_int2(0, SCORE_MIN);
    }
    
    for (int i = 0; i < rowsPadded_; i += 8) {
    
        scrUp = make_int4(gap(i), gap(i + 1), gap(i + 2), gap(i + 3));
        affUp = INT4_SCORE_MIN;
        mchUp = make_int4(gap(i - 1), gap(i), gap(i + 1), gap(i + 2));
        
        scrDown = make_int4(gap(i + 4), gap(i + 5), gap(i + 6), gap(i + 7));
        affDown = INT4_SCORE_MIN;
        mchDown = make_int4(gap(i + 3), gap(i + 4), gap(i + 5), gap(i + 6));
        
        for (int j = 0; j < cols; ++j) {
        
            int columnCodes = tex2D(seqsTexture, colOff, j + rowOff);
            
            #pragma unroll
            for (int k = 0; k < 4; ++k) {
            
                int validCol = (j * 4 + k) < realCols;
                
                wBus = hBus[(j * 4 + k) * width_ + tid];
                
                char code = (columnCodes >> (k << 3));
                char4 rowScores = tex2D(qpTexture, code, i / 4);
                
                del = max(wBus.x - gapOpen_, wBus.y - gapExtend_);
                affUp.x = max(scrUp.x - gapOpen_, affUp.x - gapExtend_);
                scrUp.x = mchUp.x + rowScores.x; 
                scrUp.x = max(scrUp.x, del);
                scrUp.x = max(scrUp.x, affUp.x);
                mchUp.x = wBus.x;
                if (i + 0 == lastRow && validCol) score = max(score, scrUp.x);
                
                del = max(scrUp.x - gapOpen_, del - gapExtend_);
                affUp.y = max(scrUp.y - gapOpen_, affUp.y - gapExtend_);
                scrUp.y = mchUp.y + rowScores.y; 
                scrUp.y = max(scrUp.y, del);
                scrUp.y = max(scrUp.y, affUp.y);
                mchUp.y = scrUp.x;
                if (i + 1 == lastRow && validCol) score = max(score, scrUp.y);
                
                del = max(scrUp.y - gapOpen_, del - gapExtend_);
                affUp.z = max(scrUp.z - gapOpen_, affUp.z - gapExtend_);
                scrUp.z = mchUp.z + rowScores.z; 
                scrUp.z = max(scrUp.z, del);
                scrUp.z = max(scrUp.z, affUp.z);
                mchUp.z = scrUp.y;
                if (i + 2 == lastRow && validCol) score = max(score, scrUp.z);
                
                del = max(scrUp.z - gapOpen_, del - gapExtend_);
                affUp.w = max(scrUp.w - gapOpen_, affUp.w - gapExtend_);
                scrUp.w = mchUp.w + rowScores.w; 
                scrUp.w = max(scrUp.w, del);
                scrUp.w = max(scrUp.w, affUp.w);
                mchUp.w = scrUp.z;
                if (i + 3 == lastRow && validCol) score = max(score, scrUp.w);

                rowScores = tex2D(qpTexture, code, i / 4 + 1);
                
                del = max(scrUp.w - gapOpen_, del - gapExtend_);
                affDown.x = max(scrDown.x - gapOpen_, affDown.x - gapExtend_);
                scrDown.x = mchDown.x + rowScores.x; 
                scrDown.x = max(scrDown.x, del);
                scrDown.x = max(scrDown.x, affDown.x);
                mchDown.x = scrUp.w;
                if (i + 4 == lastRow && validCol) score = max(score, scrDown.x);
                
                del = max(scrDown.x - gapOpen_, del - gapExtend_);
                affDown.y = max(scrDown.y - gapOpen_, affDown.y - gapExtend_);
                scrDown.y = mchDown.y + rowScores.y; 
                scrDown.y = max(scrDown.y, del);
                scrDown.y = max(scrDown.y, affDown.y);
                mchDown.y = scrDown.x;
                if (i + 5 == lastRow && validCol) score = max(score, scrDown.y);
                
                del = max(scrDown.y - gapOpen_, del - gapExtend_);
                affDown.z = max(scrDown.z - gapOpen_, affDown.z - gapExtend_);
                scrDown.z = mchDown.z + rowScores.z; 
                scrDown.z = max(scrDown.z, del);
                scrDown.z = max(scrDown.z, affDown.z);
                mchDown.z = scrDown.y;
                if (i + 6 == lastRow && validCol) score = max(score, scrDown.z);
                
                del = max(scrDown.z - gapOpen_, del - gapExtend_);
                affDown.w = max(scrDown.w - gapOpen_, affDown.w - gapExtend_);
                scrDown.w = mchDown.w + rowScores.w; 
                scrDown.w = max(scrDown.w, del);
                scrDown.w = max(scrDown.w, affDown.w);
                mchDown.w = scrDown.z;
                if (i + 7 == lastRow && validCol) score = max(score, scrDown.w);
                
                wBus.x = scrDown.w;
                wBus.y = del;
                
                hBus[(j * 4 + k) * width_ + tid] = wBus;
            }
        }
    }
    
    scores[id] = score;
}

__global__ static void nwSolveShortGpu(int* scores, int2* hBus, int* lengths, 
    int* lengthsPadded, int* offsets, int* indexes, int block) {

    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    
    if (tid + block * width_ >= length_) {
        return;
    }
    
    int id = indexes[tid + block * width_];
    int cols = lengthsPadded[id];
    int realCols = lengths[id];
    
    int colOff = id % width_;
    int rowOff = offsets[id / width_];
    
    int score = SCORE_MIN;
    
    int4 scrUp;
    int4 affUp;
    int4 mchUp;
    
    int4 scrDown;
    int4 affDown;
    int4 mchDown;
    
    int2 wBus;
    int del;
    
    int lastRow = rows_ - 1;

    for (int j = 0; j < cols * 4; ++j) {
        hBus[j * width_ + tid] = make_int2(gap(j), SCORE_MIN);
    }
    
    for (int i = 0; i < rowsPadded_; i += 8) {
    
        scrUp = make_int4(gap(i), gap(i + 1), gap(i + 2), gap(i + 3));
        affUp = INT4_SCORE_MIN;
        mchUp = make_int4(gap(i - 1), gap(i), gap(i + 1), gap(i + 2));
        
        scrDown = make_int4(gap(i + 4), gap(i + 5), gap(i + 6), gap(i + 7));
        affDown = INT4_SCORE_MIN;
        mchDown = make_int4(gap(i + 3), gap(i + 4), gap(i + 5), gap(i + 6));
        
        for (int j = 0; j < cols; ++j) {
        
            int columnCodes = tex2D(seqsTexture, colOff, j + rowOff);
            
            #pragma unroll
            for (int k = 0; k < 4; ++k) {
            
                int lastCol = (j * 4 + k) == (realCols - 1);
                
                wBus = hBus[(j * 4 + k) * width_ + tid];
                
                char code = (columnCodes >> (k << 3));
                char4 rowScores = tex2D(qpTexture, code, i / 4);
                
                del = max(wBus.x - gapOpen_, wBus.y - gapExtend_);
                affUp.x = max(scrUp.x - gapOpen_, affUp.x - gapExtend_);
                scrUp.x = mchUp.x + rowScores.x; 
                scrUp.x = max(scrUp.x, del);
                scrUp.x = max(scrUp.x, affUp.x);
                mchUp.x = wBus.x;
                if (i + 0 == lastRow && lastCol) score = scrUp.x;
                
                del = max(scrUp.x - gapOpen_, del - gapExtend_);
                affUp.y = max(scrUp.y - gapOpen_, affUp.y - gapExtend_);
                scrUp.y = mchUp.y + rowScores.y; 
                scrUp.y = max(scrUp.y, del);
                scrUp.y = max(scrUp.y, affUp.y);
                mchUp.y = scrUp.x;
                if (i + 1 == lastRow && lastCol) score = scrUp.y;
                
                del = max(scrUp.y - gapOpen_, del - gapExtend_);
                affUp.z = max(scrUp.z - gapOpen_, affUp.z - gapExtend_);
                scrUp.z = mchUp.z + rowScores.z; 
                scrUp.z = max(scrUp.z, del);
                scrUp.z = max(scrUp.z, affUp.z);
                mchUp.z = scrUp.y;
                if (i + 2 == lastRow && lastCol) score = scrUp.z;
                
                del = max(scrUp.z - gapOpen_, del - gapExtend_);
                affUp.w = max(scrUp.w - gapOpen_, affUp.w - gapExtend_);
                scrUp.w = mchUp.w + rowScores.w; 
                scrUp.w = max(scrUp.w, del);
                scrUp.w = max(scrUp.w, affUp.w);
                mchUp.w = scrUp.z;
                if (i + 3 == lastRow && lastCol) score = scrUp.w;

                rowScores = tex2D(qpTexture, code, i / 4 + 1);
                
                del = max(scrUp.w - gapOpen_, del - gapExtend_);
                affDown.x = max(scrDown.x - gapOpen_, affDown.x - gapExtend_);
                scrDown.x = mchDown.x + rowScores.x; 
                scrDown.x = max(scrDown.x, del);
                scrDown.x = max(scrDown.x, affDown.x);
                mchDown.x = scrUp.w;
                if (i + 4 == lastRow && lastCol) score = scrDown.x;
                
                del = max(scrDown.x - gapOpen_, del - gapExtend_);
                affDown.y = max(scrDown.y - gapOpen_, affDown.y - gapExtend_);
                scrDown.y = mchDown.y + rowScores.y; 
                scrDown.y = max(scrDown.y, del);
                scrDown.y = max(scrDown.y, affDown.y);
                mchDown.y = scrDown.x;
                if (i + 5 == lastRow && lastCol) score = scrDown.y;
                
                del = max(scrDown.y - gapOpen_, del - gapExtend_);
                affDown.z = max(scrDown.z - gapOpen_, affDown.z - gapExtend_);
                scrDown.z = mchDown.z + rowScores.z; 
                scrDown.z = max(scrDown.z, del);
                scrDown.z = max(scrDown.z, affDown.z);
                mchDown.z = scrDown.y;
                if (i + 6 == lastRow && lastCol) score = scrDown.z;
                
                del = max(scrDown.z - gapOpen_, del - gapExtend_);
                affDown.w = max(scrDown.w - gapOpen_, affDown.w - gapExtend_);
                scrDown.w = mchDown.w + rowScores.w; 
                scrDown.w = max(scrDown.w, del);
                scrDown.w = max(scrDown.w, affDown.w);
                mchDown.w = scrDown.z;
                if (i + 7 == lastRow && lastCol) score = scrDown.w;
                
                wBus.x = scrDown.w;
                wBus.y = del;
                
                hBus[(j * 4 + k) * width_ + tid] = wBus;
            }
        }
    }
    
    scores[id] = score;
}

__global__ static void ovSolveShortGpu(int* scores, int2* hBus, int* lengths, 
    int* lengthsPadded, int* offsets, int* indexes, int block) {

    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    
    if (tid + block * width_ >= length_) {
        return;
    }
    
    int id = indexes[tid + block * width_];
    int cols = lengthsPadded[id];
    int realCols = lengths[id];
    
    int colOff = id % width_;
    int rowOff = offsets[id / width_];
    
    int score = SCORE_MIN;
    
    int4 scrUp;
    int4 affUp;
    int4 mchUp;
    
    int4 scrDown;
    int4 affDown;
    int4 mchDown;
    
    int2 wBus;
    int del;
    
    int lastRow = rows_ - 1;
    
    for (int j = 0; j < cols * 4; ++j) {
        hBus[j * width_ + tid] = make_int2(0, SCORE_MIN);
    }
    
    for (int i = 0; i < rowsPadded_; i += 8) {
    
        scrUp = INT4_ZERO;
        affUp = INT4_SCORE_MIN;
        mchUp = INT4_ZERO;
        
        scrDown = INT4_ZERO;
        affDown = INT4_SCORE_MIN;
        mchDown = INT4_ZERO;
        
        for (int j = 0; j < cols; ++j) {
        
            int columnCodes = tex2D(seqsTexture, colOff, j + rowOff);
            
            #pragma unroll
            for (int k = 0; k < 4; ++k) {
            
                int lastCol = (j * 4 + k) == (realCols - 1);
                
                wBus = hBus[(j * 4 + k) * width_ + tid];
                
                char code = (columnCodes >> (k << 3));
                char4 rowScores = tex2D(qpTexture, code, i / 4);
                
                del = max(wBus.x - gapOpen_, wBus.y - gapExtend_);
                affUp.x = max(scrUp.x - gapOpen_, affUp.x - gapExtend_);
                scrUp.x = mchUp.x + rowScores.x; 
                scrUp.x = max(scrUp.x, del);
                scrUp.x = max(scrUp.x, affUp.x);
                mchUp.x = wBus.x;
                if (i + 0 == lastRow || lastCol) score = max(score, scrUp.x);
                
                del = max(scrUp.x - gapOpen_, del - gapExtend_);
                affUp.y = max(scrUp.y - gapOpen_, affUp.y - gapExtend_);
                scrUp.y = mchUp.y + rowScores.y; 
                scrUp.y = max(scrUp.y, del);
                scrUp.y = max(scrUp.y, affUp.y);
                mchUp.y = scrUp.x;
                if (i + 1 == lastRow || lastCol) score = max(score, scrUp.y);
                
                del = max(scrUp.y - gapOpen_, del - gapExtend_);
                affUp.z = max(scrUp.z - gapOpen_, affUp.z - gapExtend_);
                scrUp.z = mchUp.z + rowScores.z; 
                scrUp.z = max(scrUp.z, del);
                scrUp.z = max(scrUp.z, affUp.z);
                mchUp.z = scrUp.y;
                if (i + 2 == lastRow || lastCol) score = max(score, scrUp.z);
                
                del = max(scrUp.z - gapOpen_, del - gapExtend_);
                affUp.w = max(scrUp.w - gapOpen_, affUp.w - gapExtend_);
                scrUp.w = mchUp.w + rowScores.w; 
                scrUp.w = max(scrUp.w, del);
                scrUp.w = max(scrUp.w, affUp.w);
                mchUp.w = scrUp.z;
                if (i + 3 == lastRow || lastCol) score = max(score, scrUp.w);

                rowScores = tex2D(qpTexture, code, i / 4 + 1);
                
                del = max(scrUp.w - gapOpen_, del - gapExtend_);
                affDown.x = max(scrDown.x - gapOpen_, affDown.x - gapExtend_);
                scrDown.x = mchDown.x + rowScores.x; 
                scrDown.x = max(scrDown.x, del);
                scrDown.x = max(scrDown.x, affDown.x);
                mchDown.x = scrUp.w;
                if (i + 4 == lastRow || lastCol) score = max(score, scrDown.x);
                
                del = max(scrDown.x - gapOpen_, del - gapExtend_);
                affDown.y = max(scrDown.y - gapOpen_, affDown.y - gapExtend_);
                scrDown.y = mchDown.y + rowScores.y; 
                scrDown.y = max(scrDown.y, del);
                scrDown.y = max(scrDown.y, affDown.y);
                mchDown.y = scrDown.x;
                if (i + 5 == lastRow || lastCol) score = max(score, scrDown.y);
                
                del = max(scrDown.y - gapOpen_, del - gapExtend_);
                affDown.z = max(scrDown.z - gapOpen_, affDown.z - gapExtend_);
                scrDown.z = mchDown.z + rowScores.z; 
                scrDown.z = max(scrDown.z, del);
                scrDown.z = max(scrDown.z, affDown.z);
                mchDown.z = scrDown.y;
                if (i + 6 == lastRow || lastCol) score = max(score, scrDown.z);
                
                del = max(scrDown.z - gapOpen_, del - gapExtend_);
                affDown.w = max(scrDown.w - gapOpen_, affDown.w - gapExtend_);
                scrDown.w = mchDown.w + rowScores.w; 
                scrDown.w = max(scrDown.w, del);
                scrDown.w = max(scrDown.w, affDown.w);
                mchDown.w = scrDown.z;
                if (i + 7 == lastRow || lastCol) score = max(score, scrDown.w);
                
                wBus.x = scrDown.w;
                wBus.y = del;
                
                hBus[(j * 4 + k) * width_ + tid] = wBus;
            }
        }
    }
    
    scores[id] = score;
}

__global__ static void swSolveShortGpu(int* scores, int2* hBus, int* lengths, 
    int* lengthsPadded, int* offsets, int* indexes, int block) {

    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    
    if (tid + block * width_ >= length_) {
        return;
    }
    
    int id = indexes[tid + block * width_];
    int cols = lengthsPadded[id];
    
    int colOff = id % width_;
    int rowOff = offsets[id / width_];
    
    int score = 0;
    
    int4 scrUp;
    int4 affUp;
    int4 mchUp;
    
    int4 scrDown;
    int4 affDown;
    int4 mchDown;
    
    int2 wBus;
    int del;
    
    for (int j = 0; j < cols * 4; ++j) {
        hBus[j * width_ + tid] = make_int2(0, SCORE_MIN);
    }
    
    for (int i = 0; i < rowsPadded_; i += 8) {
    
        scrUp = INT4_ZERO;
        affUp = INT4_SCORE_MIN;
        mchUp = INT4_ZERO;
        
        scrDown = INT4_ZERO;
        affDown = INT4_SCORE_MIN;
        mchDown = INT4_ZERO;
        
        for (int j = 0; j < cols; ++j) {
        
            int columnCodes = tex2D(seqsTexture, colOff, j + rowOff);
            
            #pragma unroll
            for (int k = 0; k < 4; ++k) {
            
                wBus = hBus[(j * 4 + k) * width_ + tid];
                
                char code = (columnCodes >> (k << 3));
                char4 rowScores = tex2D(qpTexture, code, i / 4);
                
                del = max(wBus.x - gapOpen_, wBus.y - gapExtend_);
                affUp.x = max(scrUp.x - gapOpen_, affUp.x - gapExtend_);
                scrUp.x = mchUp.x + rowScores.x; 
                scrUp.x = max(scrUp.x, del);
                scrUp.x = max(scrUp.x, affUp.x);
                scrUp.x = max(scrUp.x, 0);
                mchUp.x = wBus.x;
                score = max(score, scrUp.x);
                
                del = max(scrUp.x - gapOpen_, del - gapExtend_);
                affUp.y = max(scrUp.y - gapOpen_, affUp.y - gapExtend_);
                scrUp.y = mchUp.y + rowScores.y; 
                scrUp.y = max(scrUp.y, del);
                scrUp.y = max(scrUp.y, affUp.y);
                scrUp.y = max(scrUp.y, 0);
                mchUp.y = scrUp.x;
                score = max(score, scrUp.y);
                
                del = max(scrUp.y - gapOpen_, del - gapExtend_);
                affUp.z = max(scrUp.z - gapOpen_, affUp.z - gapExtend_);
                scrUp.z = mchUp.z + rowScores.z; 
                scrUp.z = max(scrUp.z, del);
                scrUp.z = max(scrUp.z, affUp.z);
                scrUp.z = max(scrUp.z, 0);
                mchUp.z = scrUp.y;
                score = max(score, scrUp.z);
                
                del = max(scrUp.z - gapOpen_, del - gapExtend_);
                affUp.w = max(scrUp.w - gapOpen_, affUp.w - gapExtend_);
                scrUp.w = mchUp.w + rowScores.w; 
                scrUp.w = max(scrUp.w, del);
                scrUp.w = max(scrUp.w, affUp.w);
                scrUp.w = max(scrUp.w, 0);
                mchUp.w = scrUp.z;
                score = max(score, scrUp.w);

                rowScores = tex2D(qpTexture, code, i / 4 + 1);
                
                del = max(scrUp.w - gapOpen_, del - gapExtend_);
                affDown.x = max(scrDown.x - gapOpen_, affDown.x - gapExtend_);
                scrDown.x = mchDown.x + rowScores.x; 
                scrDown.x = max(scrDown.x, del);
                scrDown.x = max(scrDown.x, affDown.x);
                scrDown.x = max(scrDown.x, 0);
                mchDown.x = scrUp.w;
                score = max(score, scrDown.x);
                
                del = max(scrDown.x - gapOpen_, del - gapExtend_);
                affDown.y = max(scrDown.y - gapOpen_, affDown.y - gapExtend_);
                scrDown.y = mchDown.y + rowScores.y; 
                scrDown.y = max(scrDown.y, del);
                scrDown.y = max(scrDown.y, affDown.y);
                scrDown.y = max(scrDown.y, 0);
                mchDown.y = scrDown.x;
                score = max(score, scrDown.y);
                
                del = max(scrDown.y - gapOpen_, del - gapExtend_);
                affDown.z = max(scrDown.z - gapOpen_, affDown.z - gapExtend_);
                scrDown.z = mchDown.z + rowScores.z; 
                scrDown.z = max(scrDown.z, del);
                scrDown.z = max(scrDown.z, affDown.z);
                scrDown.z = max(scrDown.z, 0);
                mchDown.z = scrDown.y;
                score = max(score, scrDown.z);
                
                del = max(scrDown.z - gapOpen_, del - gapExtend_);
                affDown.w = max(scrDown.w - gapOpen_, affDown.w - gapExtend_);
                scrDown.w = mchDown.w + rowScores.w; 
                scrDown.w = max(scrDown.w, del);
                scrDown.w = max(scrDown.w, affDown.w);
                scrDown.w = max(scrDown.w, 0);
                mchDown.w = scrDown.z;
                score = max(score, scrDown.w);
                
                wBus.x = scrDown.w;
                wBus.y = del;
                
                hBus[(j * 4 + k) * width_ + tid] = wBus;
            }
        }
    }
    
    scores[id] = score;
}

//------------------------------------------------------------------------------

//------------------------------------------------------------------------------
// QUERY PROFILE

static QueryProfile* createQueryProfile(Chain* query, Scorer* scorer) {

    int rows = chainGetLength(query);
    int rowsGpu = rows + (8 - rows % 8) % 8;
    
    int width = scorerGetMaxCode(scorer) + 1;
    int height = rowsGpu / 4;

    char* row = (char*) malloc(rows * sizeof(char));
    chainCopyCodes(query, row);

    size_t size = width * height * sizeof(char4);
    char4* data = (char4*) malloc(size);
    memset(data, 0, size);
    for (int i = 0; i < height; ++i) {
        for (int j = 0; j < width - 1; ++j) {
            char4 scr;
            scr.x = i * 4 + 0 >= rows ? 0 : scorerScore(scorer, row[i * 4 + 0], j);
            scr.y = i * 4 + 1 >= rows ? 0 : scorerScore(scorer, row[i * 4 + 1], j);
            scr.z = i * 4 + 2 >= rows ? 0 : scorerScore(scorer, row[i * 4 + 2], j);
            scr.w = i * 4 + 3 >= rows ? 0 : scorerScore(scorer, row[i * 4 + 3], j);
            data[i * width + j] = scr;
        }
    }
    
    free(row);
    
    QueryProfile* queryProfile = (QueryProfile*) malloc(sizeof(QueryProfile));
    queryProfile->data = data;
    queryProfile->width = width;
    queryProfile->height = height;
    queryProfile->length = rows;
    queryProfile->size = size;
    
    return queryProfile;
}

static void deleteQueryProfile(QueryProfile* queryProfile) {
    free(queryProfile->data);
    free(queryProfile);
}

static QueryProfileGpu* createQueryProfileGpu(QueryProfile* queryProfile) {

    int width = queryProfile->width;
    int height = queryProfile->height;
    
    size_t size = queryProfile->size;
    char4* data = queryProfile->data;
    hipArray* dataGpu;
    
    CUDA_SAFE_CALL(hipMallocArray(&dataGpu, &qpTexture.channelDesc, width, height)); 
    CUDA_SAFE_CALL(hipMemcpyToArray (dataGpu, 0, 0, data, size, TO_GPU));
    CUDA_SAFE_CALL(hipBindTextureToArray(qpTexture, dataGpu));
    qpTexture.addressMode[0] = hipAddressModeClamp;
    qpTexture.addressMode[1] = hipAddressModeClamp;
    qpTexture.filterMode = hipFilterModePoint;
    qpTexture.normalized = false;
    
    size_t queryProfileGpuSize = sizeof(QueryProfileGpu);
    QueryProfileGpu* queryProfileGpu = (QueryProfileGpu*) malloc(queryProfileGpuSize);
    queryProfileGpu->data = dataGpu;
    
    return queryProfileGpu;
}

static void deleteQueryProfileGpu(QueryProfileGpu* queryProfileGpu) {
    CUDA_SAFE_CALL(hipFreeArray(queryProfileGpu->data));
    CUDA_SAFE_CALL(hipUnbindTexture(qpTexture));
    free(queryProfileGpu);
}

//------------------------------------------------------------------------------

//------------------------------------------------------------------------------
// UTILS

static int int2CmpY(const void* a_, const void* b_) {

    int2 a = *((int2*) a_);
    int2 b = *((int2*) b_);
    
    return a.y - b.y;
}

//------------------------------------------------------------------------------
//******************************************************************************

#endif // __HIPCC__

