#include "hip/hip_runtime.h"
/*
swsharp - CUDA parallelized Smith Waterman with applying Hirschberg's and 
Ukkonen's algorithm and dynamic cell pruning.
Copyright (C) 2013 Matija Korpar, contributor Mile Šikić

This program is free software: you can redistribute it and/or modify
it under the terms of the GNU General Public License as published by
the Free Software Foundation, either version 3 of the License, or
(at your option) any later version.

This program is distributed in the hope that it will be useful,
but WITHOUT ANY WARRANTY; without even the implied warranty of
MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
GNU General Public License for more details.

You should have received a copy of the GNU General Public License
along with this program.  If not, see <http://www.gnu.org/licenses/>.

Contact the author by mkorpar@gmail.com.
*/

#include <math.h>
#include <stdlib.h>
#include <stdio.h>
#include <string.h>

#include "chain.h"
#include "constants.h"
#include "cuda_utils.h"
#include "scorer.h"

#include "evalue.h"

#define MAX(a, b) ((a) > (b) ? (a) : (b))

#define SCORER_CONSTANTS_LEN (sizeof(scorerConstants) / sizeof(ScorerConstants))

struct EValueParams {
    double lambda;
    double K;
    double H;
    double a;
    double C;
    double alpha;
    double sigma;
    double b;
    double beta;
    double tau;
    double G;
    double aUn;
    double alphaUn;
    long long length;
};

typedef struct ScorerConstants {
    const char* matrix;
    int gapOpen;
    int gapExtend;
    double lambda;
    double K;
    double H;
    double a;
    double C;
    double alpha;
    double sigma;
} ScorerConstants;

// lambda, k, H, a, C, Alpha, Sigma
static ScorerConstants scorerConstants[] = {
    { "BLOSUM_62", -1, -1, 0.3176, 0.134, 0.4012, 0.7916, 0.623757, 4.964660, 4.964660 },
    { "BLOSUM_62", 11, 2, 0.297, 0.082, 0.27, 1.1, 0.641766, 12.673800, 12.757600 },
    { "BLOSUM_62", 10, 2, 0.291, 0.075, 0.23, 1.3, 0.649362, 16.474000, 16.602600 },
    { "BLOSUM_62", 9, 2, 0.279, 0.058, 0.19, 1.5, 0.659245, 22.751900, 22.950000 },
    { "BLOSUM_62", 8, 2, 0.264, 0.045, 0.15, 1.8, 0.672692, 35.483800, 35.821300 },
    { "BLOSUM_62", 7, 2, 0.239, 0.027, 0.10, 2.5, 0.702056, 61.238300, 61.886000 },
    { "BLOSUM_62", 6, 2, 0.201, 0.012, 0.061, 3.3, 0.740802, 140.417000, 141.882000 },
    { "BLOSUM_62", 13, 1, 0.292, 0.071, 0.23, 1.2, 0.647715, 19.506300, 19.893100 },
    { "BLOSUM_62", 12, 1, 0.283, 0.059, 0.19, 1.5, 0.656391, 27.856200, 28.469900 },
    { "BLOSUM_62", 11, 1, 0.267, 0.041, 0.14, 1.9, 0.669720, 42.602800, 43.636200 },
    { "BLOSUM_62", 10, 1, 0.243, 0.024, 0.10, 2.5, 0.693267, 83.178700, 85.065600 },
    { "BLOSUM_62", 9, 1, 0.206, 0.010, 0.052, 4.0, 0.731887, 210.333000, 214.842000 },
};

static __constant__ int length_;
static __constant__ int queryLen_;

static __constant__ double paramsLength_;
static __constant__ double paramsLambda_;
static __constant__ double paramsK_;
static __constant__ double paramsA_;
static __constant__ double paramsB_;
static __constant__ double paramsAlpha_;
static __constant__ double paramsBeta_;
static __constant__ double paramsSigma_;
static __constant__ double paramsTau_;

//******************************************************************************
// PUBLIC

//******************************************************************************

//******************************************************************************
// PRIVATE

static void eValuesCpu(double* values, int* scores, Chain* query, 
    Chain** database, int databaseLen, EValueParams* eValueParams);

static void eValuesGpu(double* values, int* scores, Chain* query, 
    Chain** database, int databaseLen, int* cards, int cardsLen, 
    EValueParams* eValueParams);

static double calculateEValue(int score, int queryLen, int targetLen, 
    EValueParams* params);
    
#ifdef _WIN32
double erf(double x);
#endif

// With visual c++ compiler and prototypes declared cuda global memory variables
// do not work. No questions asked.
#ifndef _WIN32
__global__ static void kernel(double* values, int2* data);
#endif

//******************************************************************************

//******************************************************************************
// PUBLIC

extern EValueParams* createEValueParams(Chain** database, int databaseLen, 
    Scorer* scorer) {

    long long length = 0;
    for (int i = 0; i < databaseLen; ++i) {
        length += chainGetLength(database[i]);
    }
    
    const char* matrix = scorerGetName(scorer);
    int gapOpen = scorerGetGapOpen(scorer);
    int gapExtend = scorerGetGapExtend(scorer);
    
    double alphaUn = scorerConstants[0].alpha;
    double aUn = scorerConstants[0].a;
    double G = gapOpen + gapExtend;
    
    int index = -1;
    for (int i = 0; i < SCORER_CONSTANTS_LEN; ++i) {

        ScorerConstants entry = scorerConstants[i];
        
        if (entry.gapOpen == gapOpen && entry.gapExtend == gapExtend &&
            strncmp(entry.matrix, matrix, strlen(entry.matrix)) == 0) {
            index = i;
            break;
        }
    }
    
    if (index == -1) {
        index = 0;
        printf("WARNING: no e-value params found, using defaults\n");
    }
    
    EValueParams* params = (EValueParams*) malloc(sizeof(struct EValueParams));
    
    params->G = G;
    params->aUn = aUn;
    params->alphaUn = alphaUn;
    params->lambda = scorerConstants[index].lambda;
    params->K = scorerConstants[index].K;
    params->H = scorerConstants[index].H;
    params->a = scorerConstants[index].a;
    params->C = scorerConstants[index].C;
    params->alpha = scorerConstants[index].alpha;
    params->sigma = scorerConstants[index].sigma;
    params->b = 2.0 * G * (params->aUn - params->a);
    params->beta = 2.0 * G * (params->alphaUn - params->alpha);
    params->tau = 2.0 * G * (params->alphaUn - params->sigma);
    params->length = length;
    
    return params;
}

extern void deleteEValueParams(EValueParams* eValueParams) {
    free(eValueParams);
    eValueParams = NULL;
}

extern void eValues(double* values, int* scores, Chain* query, 
    Chain** database, int databaseLen, int* cards, int cardsLen, 
    EValueParams* eValueParams) {

    if (cardsLen == 0) {
        eValuesCpu(values, scores, query, database, databaseLen, eValueParams);
    } else {
        eValuesGpu(values, scores, query, database, databaseLen, cards, 
            cardsLen, eValueParams);
    }
}

//******************************************************************************

//******************************************************************************
// PRIVATE

static void eValuesCpu(double* values, int* scores, Chain* query, 
    Chain** database, int databaseLen, EValueParams* eValueParams) {

    int queryLen = chainGetLength(query);

    for (int i = 0; i < databaseLen; ++i) {
        
        int score = scores[i];
        int targetLen = chainGetLength(database[i]);

        if (score == NO_SCORE) {
            values[i] = INFINITY;
            continue;
        }
        
        values[i] = calculateEValue(score, queryLen, targetLen, eValueParams);
    }
}

static void eValuesGpu(double* values, int* scores, Chain* query, 
    Chain** database, int databaseLen, int* cards, int cardsLen, 
    EValueParams* params) {

    // init cpu
    size_t dataSize = databaseLen * sizeof(int2);
    int2* dataCpu = (int2*) malloc(dataSize);
    for (int i = 0; i < databaseLen; ++i) {
        dataCpu[i].x = scores[i];
        dataCpu[i].y = chainGetLength(database[i]);
    }

    // init global memory
    size_t valuesSize = databaseLen * sizeof(double);
    double* valuesGpu;
    CUDA_SAFE_CALL(hipMalloc(&valuesGpu, valuesSize));

    int2* dataGpu;
    CUDA_SAFE_CALL(hipMalloc(&dataGpu, dataSize));
    CUDA_SAFE_CALL(hipMemcpy(dataGpu, dataCpu, dataSize, TO_GPU));

    // init constants
    int queryLen = chainGetLength(query);
    double length = params->length;

    CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(length_), &databaseLen, sizeof(int)));
    CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(queryLen_), &queryLen, sizeof(int)));

    CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(paramsLength_), &length, sizeof(double)));
    CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(paramsLambda_), &(params->lambda), sizeof(double)));
    CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(paramsK_), &(params->K), sizeof(double)));
    CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(paramsA_), &(params->a), sizeof(double)));
    CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(paramsB_), &(params->b), sizeof(double)));
    CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(paramsAlpha_), &(params->alpha), sizeof(double)));
    CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(paramsBeta_), &(params->beta), sizeof(double)));
    CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(paramsSigma_), &(params->sigma), sizeof(double)));
    CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(paramsTau_), &(params->tau), sizeof(double)));

    // solve
    kernel<<<120, 128>>>(valuesGpu, dataGpu);

    // save results
    CUDA_SAFE_CALL(hipMemcpy(values, valuesGpu, valuesSize, FROM_GPU));

    // clear memory
    CUDA_SAFE_CALL(hipFree(valuesGpu));
    CUDA_SAFE_CALL(hipFree(dataGpu));

    free(dataCpu);
}

//------------------------------------------------------------------------------
// GPU MODULES

__global__ static void kernel(double* values, int2* data) {

    int idx = blockDim.x * blockIdx.x + threadIdx.x;

    int m_ = queryLen_;

    while (idx < length_) {

        int y_ = data[idx].x;
        int n_ = data[idx].y;

        if (y_ == NO_SCORE) {
            values[idx] = INFINITY;
        } else {

            double db_scale_factor = (double) paramsLength_ / (double) n_;

            double lambda_    = paramsLambda_;
            double k_         = paramsK_;
            double ai_hat_    = paramsA_;
            double bi_hat_    = paramsB_;
            double alphai_hat_= paramsAlpha_;
            double betai_hat_ = paramsBeta_;
            double sigma_hat_ = paramsSigma_;
            double tau_hat_   = paramsTau_;

            // here we consider symmetric matrix only
            double aj_hat_    = ai_hat_;
            double bj_hat_    = bi_hat_;
            double alphaj_hat_= alphai_hat_;
            double betaj_hat_ = betai_hat_;

            // this is 1/sqrt(2.0*PI)
            double const_val = 0.39894228040143267793994605993438;
            double m_li_y, vi_y, sqrt_vi_y, m_F, P_m_F;
            double n_lj_y, vj_y, sqrt_vj_y, n_F, P_n_F;
            double c_y, p1, p2;
            double area;

            m_li_y = m_ - (ai_hat_*y_ + bi_hat_);
            vi_y = MAX(2.0*alphai_hat_/lambda_, alphai_hat_*y_+betai_hat_);
            sqrt_vi_y = sqrt(vi_y);
            m_F = m_li_y/sqrt_vi_y;
            P_m_F = 0.5 + 0.5 * erf(m_F);
            p1 = m_li_y * P_m_F + sqrt_vi_y * const_val * exp(-0.5*m_F*m_F);

            n_lj_y = n_ - (aj_hat_*y_ + bj_hat_);
            vj_y = MAX(2.0*alphaj_hat_/lambda_, alphaj_hat_*y_+betaj_hat_);
            sqrt_vj_y = sqrt(vj_y);
            n_F = n_lj_y/sqrt_vj_y;
            P_n_F = 0.5 + 0.5 * erf(n_F);
            p2 = n_lj_y * P_n_F + sqrt_vj_y * const_val * exp(-0.5*n_F*n_F);

            c_y = MAX(2.0*sigma_hat_/lambda_, sigma_hat_*y_+tau_hat_);
            area = p1 * p2 + c_y * P_m_F * P_n_F;

            values[idx] = area * k_ * exp(-lambda_ * y_) * db_scale_factor;
        }

        idx += gridDim.x * blockDim.x;
    }
}

//------------------------------------------------------------------------------

//------------------------------------------------------------------------------
// CPU MODULES

static double calculateEValue(int score, int queryLen, int targetLen, 
    EValueParams* params) {
    
    // code taken from blast
    // pile of statistical crap

    int y_ = score;
    int m_ = queryLen;
    int n_ = targetLen;
    
    // the pair-wise e-value must be scaled back to db-wise e-value
    double db_scale_factor = (double) params->length / (double) n_;

    double lambda_    = params->lambda;
    double k_         = params->K;
    double ai_hat_    = params->a;
    double bi_hat_    = params->b;
    double alphai_hat_= params->alpha;
    double betai_hat_ = params->beta;
    double sigma_hat_ = params->sigma;
    double tau_hat_   = params->tau;

    // here we consider symmetric matrix only
    double aj_hat_    = ai_hat_;
    double bj_hat_    = bi_hat_;
    double alphaj_hat_= alphai_hat_;
    double betaj_hat_ = betai_hat_;

    // this is 1/sqrt(2.0*PI)
    static double const_val = 0.39894228040143267793994605993438;
    double m_li_y, vi_y, sqrt_vi_y, m_F, P_m_F;
    double n_lj_y, vj_y, sqrt_vj_y, n_F, P_n_F;
    double c_y, p1, p2;
    double area;

    m_li_y = m_ - (ai_hat_*y_ + bi_hat_);
    vi_y = MAX(2.0*alphai_hat_/lambda_, alphai_hat_*y_+betai_hat_);
    sqrt_vi_y = sqrt(vi_y);
    m_F = m_li_y/sqrt_vi_y;
    P_m_F = 0.5 + 0.5 * erf(m_F);
    p1 = m_li_y * P_m_F + sqrt_vi_y * const_val * exp(-0.5*m_F*m_F);

    n_lj_y = n_ - (aj_hat_*y_ + bj_hat_);
    vj_y = MAX(2.0*alphaj_hat_/lambda_, alphaj_hat_*y_+betaj_hat_);
    sqrt_vj_y = sqrt(vj_y);
    n_F = n_lj_y/sqrt_vj_y;
    P_n_F = 0.5 + 0.5 * erf(n_F);
    p2 = n_lj_y * P_n_F + sqrt_vj_y * const_val * exp(-0.5*n_F*n_F);

    c_y = MAX(2.0*sigma_hat_/lambda_, sigma_hat_*y_+tau_hat_);
    area = p1 * p2 + c_y * P_m_F * P_n_F;

    return area * k_ * exp(-lambda_ * y_) * db_scale_factor;
}

#ifdef _WIN32
double erf(double x) {

    // constants
    double a1 =  0.254829592;
    double a2 = -0.284496736;
    double a3 =  1.421413741;
    double a4 = -1.453152027;
    double a5 =  1.061405429;
    double p  =  0.3275911;

    // Save the sign of x
    int sign = x < 0 ? -1 : 1;
    x = fabs(x);

    // A&S formula 7.1.26
    double t = 1.0 / (1.0 + p * x);
    double y = 1.0 - (((((a5 * t + a4)*t) + a3)*t + a2)*t + a1)*t*exp(-x*x);

    return sign * y;
}
#endif
//------------------------------------------------------------------------------
//******************************************************************************
